#include "hip/hip_runtime.h"
#include <config.h>

#include <iostream>
#include <random>
#include <primitiv/cuda_device.h>
#include <primitiv/error.h>

using std::cerr;
using std::endl;

namespace {

/**
 * Retrieves cuBLAS error string.
 * @param err cuBLAS error cude.
 * @return Error string.
 */
std::string cublasGetErrorString(::hipblasStatus_t err) {
#define MESSAGE(status) if (err == status) return #status
  MESSAGE(HIPBLAS_STATUS_SUCCESS);
  MESSAGE(HIPBLAS_STATUS_NOT_INITIALIZED);
  MESSAGE(HIPBLAS_STATUS_ALLOC_FAILED);
  MESSAGE(HIPBLAS_STATUS_INVALID_VALUE);
  MESSAGE(HIPBLAS_STATUS_ARCH_MISMATCH);
  MESSAGE(HIPBLAS_STATUS_MAPPING_ERROR);
  MESSAGE(HIPBLAS_STATUS_EXECUTION_FAILED);
  MESSAGE(HIPBLAS_STATUS_INTERNAL_ERROR);
  MESSAGE(HIPBLAS_STATUS_NOT_SUPPORTED);
  MESSAGE(HIPBLAS_STATUS_UNKNOWN);
#undef MESSAGE
  return "Unknown hipblasStatus_t value.";
}

/**
 * Retrieves cuRAND error string.
 * @param err cuRAND error cude.
 * @return Error string.
 */
std::string curandGetErrorString(::hiprandStatus_t err) {
#define MESSAGE(status) if (err == status) return #status
  MESSAGE(HIPRAND_STATUS_SUCCESS);
  MESSAGE(HIPRAND_STATUS_VERSION_MISMATCH);
  MESSAGE(HIPRAND_STATUS_NOT_INITIALIZED);
  MESSAGE(HIPRAND_STATUS_ALLOCATION_FAILED);
  MESSAGE(HIPRAND_STATUS_TYPE_ERROR);
  MESSAGE(HIPRAND_STATUS_OUT_OF_RANGE);
  MESSAGE(HIPRAND_STATUS_LENGTH_NOT_MULTIPLE);
  MESSAGE(HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED);
  MESSAGE(HIPRAND_STATUS_LAUNCH_FAILURE);
  MESSAGE(HIPRAND_STATUS_PREEXISTING_FAILURE);
  MESSAGE(HIPRAND_STATUS_INITIALIZATION_FAILED);
  MESSAGE(HIPRAND_STATUS_ARCH_MISMATCH);
  MESSAGE(HIPRAND_STATUS_INTERNAL_ERROR);
#undef MESSAGE
  return "Unknown hiprandStatus_t value.";
}

}

#define CUDA_CALL(f) { \
  ::hipError_t err = (f); \
  if (err != hipSuccess) { \
    THROW_ERROR( \
        "CUDA function failed. statement: " << #f \
        << ", error: " << err \
        << ": " << ::hipGetErrorString(err)); \
  } \
}

#define CUBLAS_CALL(f) { \
  ::hipblasStatus_t err = (f); \
  if (err != HIPBLAS_STATUS_SUCCESS) { \
    THROW_ERROR( \
        "CUBLAS function failed. statement: " << #f \
        << ", error: " << err \
        << ": " << ::cublasGetErrorString(err)); \
  } \
}

#define CURAND_CALL(f) { \
  ::hiprandStatus_t err = (f); \
  if (err != HIPRAND_STATUS_SUCCESS) { \
    THROW_ERROR( \
        "CURAND function failed. statement: " << #f \
        << ", error: " << err \
        << ": " << ::curandGetErrorString(err)); \
  } \
}

namespace {

/*
 * CUDA kernels
 */

#define IDX (threadIdx.x + blockIdx.x * blockDim.x)

__global__ void dev_set_const(float *py, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = k;
}

__global__ void dev_rand_bernoulli(float *px, float p, float size) {
  const unsigned i = IDX;
  if (i < size) px[i] = (float)(px[i] <= p);
}

__global__ void dev_rand_affine(
    float *px, float shift, float scale, unsigned size) {
  const unsigned i = IDX;
  if (i < size) px[i] = px[i] * scale + shift;
}

__global__ void dev_slice(
    float *py, const float *px, unsigned span, unsigned skip, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[(i / span) * skip + (i % span)];
}

__global__ void dev_concat(
    float *py, const float *px,
    unsigned span, unsigned skip, unsigned x_size, unsigned y_size) {
  const unsigned i = IDX;
  if (i < y_size) py[(i / span) * skip + (i % span)] = px[i % x_size];
}

__global__ void dev_negate(float *py, const float *px, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = -px[i];
}

__global__ void dev_add_const(
    float *py, const float *px, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[i] + k;
}

__global__ void dev_subtract_const_l(
    float *py, const float *px, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = k - px[i];
}

__global__ void dev_subtract_const_r(
    float *py, const float *px, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[i] - k;
}

__global__ void dev_multiply_const(
    float *py, const float *px, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[i] * k;
}

__global__ void dev_divide_const_l(
    float *py, const float *px, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = k / px[i];
}

__global__ void dev_divide_const_r(
    float *py, const float *px, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[i] / k;
}

__global__ void dev_add(
    float *py, const float *pa, const float *pb,
    unsigned size, unsigned mba, unsigned mbb) {
  const unsigned i = IDX;
  const unsigned shift = blockIdx.y * size;
  if (i < size) py[i + shift] = pa[i + mba * shift] + pb[i + mbb * shift];
}

__global__ void dev_subtract(
    float *py, const float *pa, const float *pb,
    unsigned size, unsigned mba, unsigned mbb) {
  const unsigned i = IDX;
  const unsigned shift = blockIdx.y * size;
  if (i < size) py[i + shift] = pa[i + mba * shift] - pb[i + mbb * shift];
}

__global__ void dev_multiply(
    float *py, const float *pa, const float *pb,
    unsigned size, unsigned mba, unsigned mbb) {
  const unsigned i = IDX;
  const unsigned shift = blockIdx.y * size;
  if (i < size) py[i + shift] = pa[i + mba * shift] * pb[i + mbb * shift];
}

__global__ void dev_divide(
    float *py, const float *pa, const float *pb,
    unsigned size, unsigned mba, unsigned mbb) {
  const unsigned i = IDX;
  const unsigned shift = blockIdx.y * size;
  if (i < size) py[i + shift] = pa[i + mba * shift] / pb[i + mbb * shift];
}

__global__ void dev_transpose(
    float *py, const float *px, unsigned rows, unsigned cols) {
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned ofs = blockIdx.z * rows * cols;
  if (i < rows && j < cols) {
    py[ofs + j + i * cols] = px[ofs + i + j * rows];
  }
}

__global__ void dev_exp(float *py, const float *px, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = ::expf(px[i]);
}

__global__ void dev_tanh(float *py, const float *px, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = ::tanhf(px[i]);
}

__global__ void dev_sigmoid(float *py, const float *px, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = .5f + .5f * ::tanhf(.5f * px[i]);
}

__global__ void dev_step(float *py, const float *px, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = (float)(px[i] > .0f);
}

__global__ void dev_relu(float *py, const float *px, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = ::fmaxf(px[i], .0f);
}

__global__ void dev_batch_sum(
    float *py, const float *px, unsigned size, unsigned batch) {
  const unsigned i = IDX;
  if (i < size) {
    float temp = .0f;
    px += i;
    for (unsigned j = 0; j < batch; ++j, px += size) {
      temp += *px;
    }
    py[i] = temp;
  }
}

__global__ void dev_add_grad(
    float *pgx, const float *pgy, unsigned nx, unsigned ny) {
  const unsigned i = IDX;
  if (i < ::max(nx, ny)) ::atomicAdd(pgx + i % nx, pgy[i % ny]);
}

__global__ void dev_add_grad_ofs(
    float *pgx, const float *pgy,
    unsigned wx, unsigned wy, unsigned nx, unsigned ny) {
  const unsigned i = IDX;
  if (i < wy * ::max(nx, ny)) {
    ::atomicAdd(
        pgx + ((i / wy) * wx + (i % wy)) % (wx * nx),
        pgy[i % (wy * ny)]);
  }
}

#undef IDX

}  // namespace

namespace primitiv {

void CUDADevice::initialize() {
  // Retrieves device properties.
  int max_devs;
  CUDA_CALL(::hipGetDeviceCount(&max_devs));
  if (dev_id_ >= static_cast<unsigned>(max_devs)) {
    THROW_ERROR(
        "Invalid CUDA device ID. given: " << dev_id_ << " >= " << max_devs);
  }
  CUDA_CALL(::hipGetDeviceProperties(&prop_, dev_id_));

  // Dump device properties.
  cerr << "Selected CUDA Device " << dev_id_ << ':' << endl;
  cerr << "  Name ............ " << prop_.name << endl;
  cerr << "  Global Memory ... " << prop_.totalGlobalMem << endl;
  cerr << "  Shared Memory ... " << prop_.sharedMemPerBlock << endl;
  cerr << "  Threads/block ... " << prop_.maxThreadsPerBlock << endl;
  cerr << "  Threads dim ..... " << prop_.maxThreadsDim[0] << ','
                                 << prop_.maxThreadsDim[1] << ','
                                 << prop_.maxThreadsDim[2] << endl;
  cerr << "  Grid size ....... " << prop_.maxGridSize[0] << ','
                                 << prop_.maxGridSize[1] << ','
                                 << prop_.maxGridSize[2] << endl;

  // Calculates size of dims to be used in CUDA kernels.
  dim1_x_ = dim2_y_ = prop_.maxThreadsPerBlock;
  dim2_x_ = 1;
  while (dim2_x_ < dim2_y_) {
    dim2_x_ <<= 1;
    dim2_y_ >>= 1;
  }
  cerr << "Block configuration:" << endl;
  cerr << "  1 dim .... " << dim1_x_ << " threads" << endl;
  cerr << "  2 dims ... " << dim2_x_ << "x" << dim2_y_ << " threads" << endl;

  // Additional libraries
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUBLAS_CALL(::hipblasCreate(&cublas_));
  CURAND_CALL(::hiprandCreateGenerator(&curand_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(::hiprandSetPseudoRandomGeneratorSeed(curand_, rng_seed_));
}

CUDADevice::CUDADevice(unsigned device_id)
: dev_id_(device_id)
, rng_seed_(std::random_device()()) {
  initialize();
}

CUDADevice::CUDADevice(unsigned device_id, unsigned rng_seed)
: dev_id_(device_id)
, rng_seed_(rng_seed) {
  initialize();
}

CUDADevice::~CUDADevice() {
  // Check memory leak
  if (!blocks_.empty()) {
    cerr << "FATAL ERROR: Detected memory leak on CUDADevice!" << endl;
    cerr << "Leaked blocks (handle: size):" << endl;
    for (const auto &kv : blocks_) {
      cerr << "  " << kv.first << ": " << kv.second << endl;
    }
    std::abort();
  }

  // Additional libraries
  CUBLAS_CALL(::hipblasDestroy(cublas_));
  CURAND_CALL(::hiprandDestroyGenerator(curand_));
}

void *CUDADevice::new_handle(const Shape &shape) {
  const unsigned mem_size = sizeof(float) * shape.size();
  void *data;
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMalloc(&data, mem_size));
  blocks_.insert(std::make_pair(data, mem_size));
  return data;
}

void CUDADevice::delete_tensor_impl(Tensor &x) {
  void *data = x.data();
  auto it = blocks_.find(data);
  if (it == blocks_.end()) {
    THROW_ERROR("Attempted to dispose unknown memory block: " << data);
  }
  blocks_.erase(it);
  CUDA_CALL(::hipFree(data));
}

#define GRID_SIZE(x, threads) (((x) + (threads) - 1) / (threads))
#define DATA(x) static_cast<float *>((x).data())
#define CDATA(x) static_cast<const float *>((x).data())

std::vector<float> CUDADevice::tensor_to_vector_impl(const Tensor &x) {
  const unsigned size = x.shape().size();
  std::vector<float> ret(size);
  CUDA_CALL(::hipMemcpy(
        &ret[0], x.data(), sizeof(float) * size, hipMemcpyDeviceToHost));
  return ret;
}

void CUDADevice::reset_tensor_impl(Tensor &x, float k) {
  const unsigned size = x.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  ::dev_set_const<<<num_blocks, dim1_x_>>>(DATA(x), k, size);
}

void CUDADevice::reset_tensor_impl(
    Tensor &x, const std::vector<float> &values) {
  const unsigned size = x.shape().size();
  CUDA_CALL(::hipMemcpy(
        x.data(), &values[0], sizeof(float) * size, hipMemcpyHostToDevice));
}

Tensor CUDADevice::random_bernoulli_impl(const Shape &shape, float p) {
  const unsigned size = shape.size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  Tensor ret = new_tensor(shape);
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(ret), size));
  ::dev_rand_bernoulli<<<num_blocks, dim1_x_>>>(DATA(ret), p, size);
  return ret;
}

Tensor CUDADevice::random_uniform_impl(
    const Shape &shape, float lower, float upper) {
  const unsigned size = shape.size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  const float scale = upper - lower;
  Tensor ret = new_tensor(shape);
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(ret), size));
  ::dev_rand_affine<<<num_blocks, dim1_x_>>>(DATA(ret), lower, scale, size);
  return ret;
}

Tensor CUDADevice::random_normal_impl(
    const Shape &shape, float mean, float sd) {
  const unsigned size = shape.size();
  Tensor ret = new_tensor(shape);
  CURAND_CALL(::hiprandGenerateNormal(curand_, DATA(ret), size, mean, sd));
  return ret;
}

Tensor CUDADevice::slice_impl(
    const Tensor &x, unsigned dim, unsigned offset, const Shape &new_shape) {
  unsigned base = 1;
  for (unsigned i = 0; i < dim; ++i) base *= new_shape.dim(i);
  const unsigned span = base * new_shape.dim(dim);
  const unsigned skip = base * x.shape().dim(dim);
  const unsigned size = new_shape.size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  Tensor ret = new_tensor(new_shape);
  ::dev_slice<<<num_blocks, dim1_x_>>>(
      DATA(ret), CDATA(x) + base * offset, span, skip, size);
  return ret;
}

Tensor CUDADevice::concat_impl(
    const std::vector<const Tensor *> &xs,
    unsigned dim, const Shape &new_shape) {
  const std::vector<unsigned> new_dims = new_shape.dims();
  const unsigned new_bs = new_shape.batch_size();
  unsigned base = 1;
  for (unsigned i = 0; i < dim; ++i) base *= new_dims[i];
  unsigned repeat = 1;
  for (unsigned i = dim + 1; i < new_dims.size(); ++i) repeat *= new_dims[i];

  Tensor ret = new_tensor(new_shape);
  unsigned offset = 0;
  for (const Tensor *x : xs) {
    const unsigned span = base * x->shape().dim(dim);
    const unsigned skip = base * new_dims[dim];
    const unsigned x_size = span * repeat * x->shape().batch_size();
    const unsigned y_size = span * repeat * new_bs;
    const unsigned num_blocks = GRID_SIZE(y_size, dim1_x_);
    ::dev_concat<<<num_blocks, dim1_x_>>>(
        DATA(ret) + offset, CDATA(*x), span, skip, x_size, y_size);
    offset += span;
  }
  return ret;
}

Tensor CUDADevice::duplicate_impl(const Tensor &x) {
  Tensor ret = new_tensor(x.shape());
  CUDA_CALL(::hipMemcpy(
      ret.data(), x.data(), sizeof(float) * x.shape().size(),
      hipMemcpyDeviceToDevice));
  return ret;
}

#define CUDA_DEV_UNARY(name, kernel) \
Tensor CUDADevice::name(const Tensor &x) { \
  Tensor ret = new_tensor(x.shape()); \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_); \
  ::kernel<<<num_blocks, dim1_x_>>>(DATA(ret), CDATA(x), size); \
  return ret; \
}

#define CUDA_DEV_BINARY_KX(name, kernel) \
Tensor CUDADevice::name(float k, const Tensor &x) { \
  Tensor ret = new_tensor(x.shape()); \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_); \
  ::kernel<<<num_blocks, dim1_x_>>>(DATA(ret), CDATA(x), k, size); \
  return ret; \
}

#define CUDA_DEV_BINARY_XK(name, kernel) \
Tensor CUDADevice::name(const Tensor &x, float k) { \
  Tensor ret = new_tensor(x.shape()); \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size,dim1_x_); \
  ::kernel<<<num_blocks, dim1_x_>>>(DATA(ret), CDATA(x), k, size); \
  return ret; \
}

#define CUDA_DEV_BINARY_AB(name, kernel) \
Tensor CUDADevice::name(const Tensor &a, const Tensor &b) { \
  const Shape &sa = a.shape(); \
  const Shape &sb = b.shape(); \
  const unsigned ba = sa.batch_size(); \
  const unsigned bb = sb.batch_size(); \
  const unsigned size = sa.size() / ba; \
  const unsigned x = GRID_SIZE(size, dim1_x_); \
  const unsigned y = std::max(ba, bb); \
  Tensor ret = new_tensor(Shape(sa.dims(), y)); \
  ::kernel<<<dim3(x, y, 1), dim1_x_>>>( \
      DATA(ret), CDATA(a), CDATA(b), size, ba > 1, bb > 1); \
  return ret; \
}

CUDA_DEV_UNARY(negate_impl, dev_negate);
CUDA_DEV_UNARY(exp_impl, dev_exp);
CUDA_DEV_UNARY(tanh_impl, dev_tanh);
CUDA_DEV_UNARY(sigmoid_impl, dev_sigmoid);
CUDA_DEV_UNARY(step_impl, dev_step);
CUDA_DEV_UNARY(relu_impl, dev_relu);

CUDA_DEV_BINARY_XK(add_impl, dev_add_const);
CUDA_DEV_BINARY_KX(subtract_impl, dev_subtract_const_l);
CUDA_DEV_BINARY_XK(subtract_impl, dev_subtract_const_r);
CUDA_DEV_BINARY_XK(multiply_impl, dev_multiply_const);
CUDA_DEV_BINARY_KX(divide_impl, dev_divide_const_l);
CUDA_DEV_BINARY_XK(divide_impl, dev_divide_const_r);

CUDA_DEV_BINARY_AB(add_impl, dev_add);
CUDA_DEV_BINARY_AB(subtract_impl, dev_subtract);
CUDA_DEV_BINARY_AB(multiply_impl, dev_multiply);
CUDA_DEV_BINARY_AB(divide_impl, dev_divide);

#undef CUDA_DEV_UNARY
#undef CUDA_DEV_BINARY_KX
#undef CUDA_DEV_BINARY_XK
#undef CUDA_DEV_BINARY_AB

Tensor CUDADevice::transpose_impl(const Tensor &x) {
  const Shape &s = x.shape();
  const unsigned d1 = s.dim(0);
  const unsigned d2 = s.dim(1);
  const unsigned bs = s.batch_size();
  const unsigned g1 = GRID_SIZE(d1, dim2_x_);
  const unsigned g2 = GRID_SIZE(d2, dim2_y_);
  Tensor ret = new_tensor(Shape({d2, d1}, bs));
  ::dev_transpose<<<dim3(g1, g2, bs), dim3(dim2_x_, dim2_y_, 1)>>>(
      DATA(ret), CDATA(x), d1, d2);
  return ret;
}

Tensor CUDADevice::dot_impl(const Tensor &a, const Tensor &b) {
  const Shape &sa = a.shape();
  const Shape &sb = b.shape();
  const unsigned di = sa.dim(0);
  const unsigned dj = sa.dim(1);
  const unsigned dk = sb.dim(1);
  const unsigned ba = sa.batch_size();
  const unsigned bb = sb.batch_size();
  const unsigned bs = std::max(ba, bb);
  float alpha = 1.;
  float beta = 0.;
  Tensor ret = new_tensor(Shape({di, dk}, bs));
  reset_tensor(ret, 0);
  if (ba == 1) {
    // Do gemm only once to calculate dot with combined matrices.
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
          di, bb * dk, dj,
          &alpha, CDATA(a), di, CDATA(b), dj,
          &beta, DATA(ret), di));
  } else {
    // Do gemm multiple times.
    const unsigned a_skip = di * dj;
    const unsigned b_skip = static_cast<unsigned>(bb > 1) * dj * dk;
    const unsigned y_skip = di * dk;
    for (unsigned n = 0; n < ba; ++n) {
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
            di, dk, dj,
            &alpha, CDATA(a) + n * a_skip, di, CDATA(b) + n * b_skip, dj,
            &beta, DATA(ret) + n * y_skip, di));
    }
  }
  return ret;
}


Tensor CUDADevice::batch_sum_impl(const Tensor &x) {
  Tensor ret = new_tensor(Shape(x.shape().dims()));
  const unsigned size = ret.shape().size();
  const unsigned g1 = GRID_SIZE(size, dim1_x_);
  ::dev_batch_sum<<<g1, dim1_x_>>>(
      DATA(ret), CDATA(x), size, x.shape().batch_size());
  return ret;
}

void CUDADevice::add_gradient_impl(Tensor &a, const Tensor &b) {
  const unsigned nx = a.shape().size();
  const unsigned ny = b.shape().size();
  const unsigned g1 = GRID_SIZE(std::max(nx, ny), dim1_x_);
  ::dev_add_grad<<<g1, dim1_x_>>>(DATA(a), CDATA(b), nx, ny);
}

void CUDADevice::add_gradient_offset_impl(
    Tensor &a, const Tensor &b, unsigned dim, unsigned offset) {
  const Shape &sa = a.shape();
  const Shape &sb = b.shape();
  unsigned base = 1;
  for (unsigned i = 0; i < dim; ++i) base *= sa.dim(i);
  unsigned repeat = 1;
  for (unsigned i = dim + 1; i < sa.dims().size(); ++i) repeat *= sa.dim(i);
  const unsigned ox = base * offset;
  const unsigned wx = base * sa.dim(dim);
  const unsigned wy = base * sb.dim(dim);
  const unsigned nx = repeat * sa.batch_size();
  const unsigned ny = repeat * sb.batch_size();
  const unsigned g1 = GRID_SIZE(wy * std::max(nx, ny), dim1_x_);
  ::dev_add_grad_ofs<<<g1, dim1_x_>>>(DATA(a) + ox, CDATA(b), wx, wy, nx, ny);
}

}  // namespace primitiv
