#include "hip/hip_runtime.h"
#include <config.h>

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <random>
#include <primitiv/cuda_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/error.h>

using std::cerr;
using std::endl;

namespace {

/*
 * CUDA kernels
 */

#define IDX (threadIdx.x + blockIdx.x * blockDim.x)

__global__ void set_const_dev(float *py, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = k;
}

__global__ void rand_bernoulli_dev(float *px, float p, float size) {
  const unsigned i = IDX;
  if (i < size) px[i] = (float)(px[i] <= p);
}

__global__ void rand_affine_dev(
    float *px, float shift, float scale, unsigned size) {
  const unsigned i = IDX;
  if (i < size) px[i] = px[i] * scale + shift;
}

__global__ void slice_fw_dev(
    float *py, const float *px, unsigned span, unsigned skip, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[(i / span) * skip + (i % span)];
}

__global__ void concat_fw_dev(
    float *py, const float *px,
    unsigned span, unsigned skip, unsigned x_size, unsigned y_size) {
  const unsigned i = IDX;
  if (i < y_size) py[(i / span) * skip + (i % span)] = px[i % x_size];
}

#define CUDA_KERNEL_X(name, op) \
__global__ void name##_fw_dev(float *py, const float *px, unsigned size) { \
  const unsigned i = IDX; \
  if (i < size) py[i] = (op); \
}

#define CUDA_KERNEL_X_CONST(name, op) \
__global__ void name##_fw_dev(float *py, const float *px, float k, unsigned size) { \
  const unsigned i = IDX; \
  if (i < size) py[i] = (op); \
}

#define CUDA_KERNEL_X_SCALAR_R(name, op) \
__global__ void name##_fw_dev( \
    float *py, const float *px, const float *pk, \
    unsigned size, unsigned mbx, unsigned mbk) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(px[i + mbx * shift], pk[mbk * blockIdx.y]); \
}

#define CUDA_KERNEL_X_SCALAR_L(name, op) \
__global__ void name##_fw_dev( \
    float *py, const float *px, const float *pk, \
    unsigned size, unsigned mbx, unsigned mbk) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(pk[mbk * blockIdx.y], px[i + mbx * shift]); \
}

#define CUDA_KERNEL_AB(name, op) \
__global__ void name##_fw_dev( \
    float *py, const float *pa, const float *pb, \
    unsigned size, unsigned mba, unsigned mbb) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(pa[i + mba * shift], pb[i + mbb * shift]); \
}

CUDA_KERNEL_X(negate, -px[i]);
CUDA_KERNEL_X(sqrt, ::__fsqrt_rn(px[i]));
CUDA_KERNEL_X(exp, ::expf(px[i]));
CUDA_KERNEL_X(tanh, ::tanhf(px[i]));
CUDA_KERNEL_X(sigmoid, .5f + .5f * ::tanhf(.5f * px[i]));
CUDA_KERNEL_X(sin, ::sinf(px[i]));
CUDA_KERNEL_X(cos, ::cosf(px[i]));
CUDA_KERNEL_X(tan, ::tanf(px[i]));

CUDA_KERNEL_X_CONST(add_const, px[i] + k);
CUDA_KERNEL_X_CONST(subtract_const_r, px[i] - k);
CUDA_KERNEL_X_CONST(subtract_const_l, k - px[i]);
CUDA_KERNEL_X_CONST(multiply_const, px[i] * k);
CUDA_KERNEL_X_CONST(divide_const_r, px[i] / k);
CUDA_KERNEL_X_CONST(divide_const_l, k / px[i]);
CUDA_KERNEL_X_CONST(pstep, (px[i] > .0f) + k * (px[i] <= .0f));
CUDA_KERNEL_X_CONST(prelu, px[i] * ((px[i] > .0f) + k * (px[i] <= .0f)));

CUDA_KERNEL_X_SCALAR_R(add_scalar, ::__fadd_rn);
CUDA_KERNEL_X_SCALAR_R(subtract_scalar_r, ::__fsub_rn);
CUDA_KERNEL_X_SCALAR_L(subtract_scalar_l, ::__fsub_rn);
CUDA_KERNEL_X_SCALAR_R(multiply_scalar, ::__fmul_rn);
CUDA_KERNEL_X_SCALAR_R(divide_scalar_r, ::__fdiv_rn);
CUDA_KERNEL_X_SCALAR_L(divide_scalar_l, ::__fdiv_rn);

CUDA_KERNEL_AB(add, ::__fadd_rn);
CUDA_KERNEL_AB(subtract, ::__fsub_rn);
CUDA_KERNEL_AB(multiply, ::__fmul_rn);
CUDA_KERNEL_AB(divide, ::__fdiv_rn);

#undef CUDA_KERNEL_X
#undef CUDA_KERNEL_X_CONST
#undef CUDA_KERNEL_X_SCALAR_R
#undef CUDA_KERNEL_X_SCALAR_L

__global__ void transpose_fw_dev(
    float *py, const float *px, unsigned rows, unsigned cols) {
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned ofs = blockIdx.z * rows * cols;
  if (i < rows && j < cols) {
    py[ofs + j + i * cols] = px[ofs + i + j * rows];
  }
}

template<unsigned BLOCK_SIZE>
__global__ void sum_fw_dev(float *py, const float *px, unsigned skip, unsigned n) {
  __shared__ float temp[BLOCK_SIZE];
  const unsigned bid = blockIdx.x;
  const unsigned tid = threadIdx.x;
  px += bid % skip + (bid / skip) * skip * n;
  temp[tid] = 0;
  for (unsigned i = tid; i < n; i += BLOCK_SIZE) temp[tid] += px[i * skip];
  __syncthreads();
#define REDUCE(k) \
  if (BLOCK_SIZE >= k << 1) { \
    if (tid < k) temp[tid] += temp[tid + k]; \
    __syncthreads(); \
  }
  REDUCE(512)
  REDUCE(256)
  REDUCE(128)
  REDUCE(64)
  REDUCE(32)
  REDUCE(16)
  REDUCE(8)
  REDUCE(4)
  REDUCE(2)
  REDUCE(1)
#undef REDUCE
  if (tid == 0) py[bid] = temp[0];
}

__device__ float logsumexp2_fw_dev(float a, float b) {
  return a > b
    ? a + ::log(1.f + ::exp(b - a))
    : b + ::log(1.f + ::exp(a - b));
}

template<unsigned BLOCK_SIZE>
__global__ void logsumexp_fw_dev(
    float *py, const float *px, unsigned skip, unsigned n) {
  __shared__ float temp[BLOCK_SIZE];
  const unsigned bid = blockIdx.x;
  const unsigned tid = threadIdx.x;
  px += bid % skip + (bid / skip) * skip * n;
  temp[tid] = -1e38;  // NOTE(odashi): Near the minimum of the float.
  for (unsigned i = tid; i < n; i += BLOCK_SIZE) {
    temp[tid] = ::logsumexp2_fw_dev(temp[tid], px[i * skip]);
  }
  __syncthreads();
#define REDUCE(k) \
  if (BLOCK_SIZE >= k << 1) { \
    if (tid < k) temp[tid] = ::logsumexp2_fw_dev(temp[tid], temp[tid + k]); \
    __syncthreads(); \
  }
  REDUCE(512)
  REDUCE(256)
  REDUCE(128)
  REDUCE(64)
  REDUCE(32)
  REDUCE(16)
  REDUCE(8)
  REDUCE(4)
  REDUCE(2)
  REDUCE(1)
#undef REDUCE
  if (tid == 0) py[bid] = temp[0];
}

__global__ void broadcast_fw_dev(
    float *py, const float *px, unsigned skip1, unsigned skip2, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[i % skip1 + (i / skip2) * skip1];
}

__global__ void batch_sum_fw_dev(
    float *py, const float *px, unsigned size, unsigned batch) {
  const unsigned i = IDX;
  if (i < size) {
    float temp = .0f;
    px += i;
    for (unsigned j = 0; j < batch; ++j, px += size) {
      temp += *px;
    }
    py[i] = temp;
  }
}

__global__ void add_grad_dev(
    float *pgx, const float *pgy, unsigned nx, unsigned ny) {
  const unsigned i = IDX;
  if (i < ::max(nx, ny)) ::atomicAdd(pgx + i % nx, pgy[i % ny]);
}

__global__ void add_grad_ofs_dev(
    float *pgx, const float *pgy,
    unsigned wx, unsigned wy, unsigned nx, unsigned ny) {
  const unsigned i = IDX;
  if (i < wy * ::max(nx, ny)) {
    ::atomicAdd(
        pgx + ((i / wy) * wx + (i % wy)) % (wx * nx),
        pgy[i % (wy * ny)]);
  }
}

__global__ void add_grad_sparse_dev(
    float *pgx, const float *pgy, unsigned wx, unsigned wy, unsigned repeat) {
  const unsigned i = IDX;
  if (i < wy * repeat) {
    ::atomicAdd(pgx + (i / wy) * wx + (i % wy), pgy[i]);
  }
}

#undef IDX

}  // namespace

namespace {

// Minimum requirements of the compute capability.
static const int MIN_CC_MAJOR = 3;
static const int MIN_CC_MINOR = 0;

}

namespace primitiv {

unsigned CUDADevice::num_devices() {
  int ret;
  CUDA_CALL(::hipGetDeviceCount(&ret));
  return ret;
}

void CUDADevice::initialize() {
  // Retrieves device properties.
  ::hipDeviceProp_t prop;
  CUDA_CALL(::hipGetDeviceProperties(&prop, dev_id_));

  // Dump device properties.
  cerr << "Selected CUDA Device " << dev_id_ << ':' << endl;
  cerr << "  Name ................. " << prop.name << endl;
  cerr << "  Global Memory ........ " << prop.totalGlobalMem << endl;
  cerr << "  Shared Memory ........ " << prop.sharedMemPerBlock << endl;
  cerr << "  Threads/block ........ " << prop.maxThreadsPerBlock << endl;
  cerr << "  Threads dim .......... " << prop.maxThreadsDim[0] << ','
                                      << prop.maxThreadsDim[1] << ','
                                      << prop.maxThreadsDim[2] << endl;
  cerr << "  Grid size ............ " << prop.maxGridSize[0] << ','
                                      << prop.maxGridSize[1] << ','
                                      << prop.maxGridSize[2] << endl;
  cerr << "  Compute Capability ... " << prop.major << '.'
                                      << prop.minor << endl;

  // Check compute capability requirements.
  if (prop.major < ::MIN_CC_MAJOR ||
      (prop.major == ::MIN_CC_MAJOR && prop.minor < ::MIN_CC_MINOR)) {
    THROW_ERROR(
        "CUDA Device " << dev_id_ << " does not satisfy the "
        "minimum requirement of the compute capability: "
        << prop.major << '.' << prop.minor << " < "
        << ::MIN_CC_MAJOR << '.' << ::MIN_CC_MINOR);
  }

  // Calculates size of dims to be used in CUDA kernels.
  dim1_x_ = 1;
  while (dim1_x_ < 1024 &&
      dim1_x_ < static_cast<unsigned>(prop.maxThreadsPerBlock)) {
    dim1_x_ <<= 1;
  }
  dim2_y_ = dim1_x_;
  dim2_x_ = 1;
  while (dim2_x_ < dim2_y_) {
    dim2_x_ <<= 1;
    dim2_y_ >>= 1;
  }
  cerr << "Block configuration:" << endl;
  cerr << "  1 dim .... " << dim1_x_ << " threads" << endl;
  cerr << "  2 dims ... " << dim2_x_ << "x" << dim2_y_ << " threads" << endl;

  // Initializes additional libraries
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUBLAS_CALL(::hipblasCreate(&cublas_));
  CURAND_CALL(::hiprandCreateGenerator(&curand_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(::hiprandSetPseudoRandomGeneratorSeed(curand_, rng_seed_));
}

CUDADevice::CUDADevice(unsigned device_id)
: dev_id_(device_id)
, rng_seed_(std::random_device()())
, pool_(device_id) {
  initialize();
}

CUDADevice::CUDADevice(unsigned device_id, unsigned rng_seed)
: dev_id_(device_id)
, rng_seed_(rng_seed)
, pool_(device_id) {
  initialize();
}

CUDADevice::~CUDADevice() {
  // Finalizes additional libraries
  CUBLAS_CALL(::hipblasDestroy(cublas_));
  CURAND_CALL(::hiprandDestroyGenerator(curand_));
}

std::shared_ptr<void> CUDADevice::new_handle(const Shape &shape) {
  return pool_.allocate(sizeof(float) * shape.size());
}

#define GRID_SIZE(x, threads) (((x) + (threads) - 1) / (threads))
#define DATA(x) static_cast<float *>((x).data())
#define CDATA(x) static_cast<const float *>((x).data())

std::vector<float> CUDADevice::tensor_to_vector_impl(const Tensor &x) {
  const unsigned size = x.shape().size();
  std::vector<float> ret(size);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        &ret[0], x.data(), sizeof(float) * size, hipMemcpyDeviceToHost));
  return ret;
}

void CUDADevice::reset_tensor_impl(Tensor &x, float k) {
  const unsigned size = x.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_const_dev<<<num_blocks, dim1_x_>>>(DATA(x), k, size);
}

void CUDADevice::reset_tensor_by_array_impl(Tensor &x, const float values[]) {
  const unsigned size = x.shape().size();
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        x.data(), values, sizeof(float) * size, hipMemcpyHostToDevice));
}

void CUDADevice::copy_tensor_impl(const Tensor &x, Tensor &y) {
  switch (x.device()->type()) {
    case Device::DEVICE_TYPE_CPU:
      reset_tensor_by_array(y, CDATA(x));
      break;
    case Device::DEVICE_TYPE_CUDA:
      CUDA_CALL(::hipSetDevice(dev_id_));
      CUDA_CALL(::hipMemcpy(
            DATA(y), CDATA(x),
            sizeof(float) * x.shape().size(),
            hipMemcpyDeviceToDevice));
      break;
    default:
      reset_tensor_by_vector(y, x.to_vector());
  }
}

void CUDADevice::random_bernoulli_impl(float p, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(y), size));
  ::rand_bernoulli_dev<<<num_blocks, dim1_x_>>>(DATA(y), p, size);
}

void CUDADevice::random_uniform_impl(float lower, float upper, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  const float scale = upper - lower;
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(y), size));
  ::rand_affine_dev<<<num_blocks, dim1_x_>>>(DATA(y), lower, scale, size);
}

void CUDADevice::random_normal_impl(float mean, float sd, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateNormal(
        curand_, DATA(y), y.shape().size(), mean, sd));
}

void CUDADevice::random_log_normal_impl(float mean, float sd, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateLogNormal(
        curand_, DATA(y), y.shape().size(), mean, sd));
}

void CUDADevice::pick_fw_impl(
    const Tensor &x, unsigned dim,
    const std::vector<unsigned> &ids, Tensor &y) {
  const unsigned base = y.shape().lower_volume(dim);
  const unsigned skip = base * x.shape()[dim];
  const unsigned size = y.shape().volume();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  const unsigned skip_x = x.shape().has_batch() * x.shape().volume();
  const unsigned skip_i = ids.size() > 1;
  const unsigned bs = y.shape().batch();
  CUDA_CALL(::hipSetDevice(dev_id_));
  for (unsigned b = 0; b < bs; ++b) {
    ::slice_fw_dev<<<num_blocks, dim1_x_>>>(
        DATA(y) + b * size, CDATA(x) + b * skip_x + base * ids[b * skip_i],
        base, skip, size);
  }
}

void CUDADevice::slice_fw_impl(
    const Tensor &x, unsigned dim, unsigned offset, Tensor &y) {
  const unsigned base = y.shape().lower_volume(dim);
  const unsigned span = base * y.shape()[dim];
  const unsigned skip = base * x.shape()[dim];
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::slice_fw_dev<<<num_blocks, dim1_x_>>>(
      DATA(y), CDATA(x) + base * offset, span, skip, size);
}

void CUDADevice::concat_fw_impl(
    const std::vector<const Tensor *> &xs, unsigned dim, Tensor &y) {
  const unsigned new_bs = y.shape().batch();
  const unsigned base = y.shape().lower_volume(dim);
  const unsigned skip = base * y.shape()[dim];
  unsigned repeat = y.shape().volume() / skip;
  CUDA_CALL(::hipSetDevice(dev_id_));
  unsigned offset = 0;
  for (const Tensor *x : xs) {
    const unsigned span = base * x->shape()[dim];
    const unsigned x_size = span * repeat * x->shape().batch();
    const unsigned y_size = span * repeat * new_bs;
    const unsigned num_blocks = GRID_SIZE(y_size, dim1_x_);
    ::concat_fw_dev<<<num_blocks, dim1_x_>>>(
        DATA(y) + offset, CDATA(*x), span, skip, x_size, y_size);
    offset += span;
  }
}

#define CUDADEV_FW_X(name) \
void CUDADevice::name##_fw_impl(const Tensor &x, Tensor &y) { \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<num_blocks, dim1_x_>>>(DATA(y), CDATA(x), size); \
}

#define CUDADEV_FW_X_CONST(name) \
void CUDADevice::name##_fw_impl(const Tensor &x, float k, Tensor &y) { \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size,dim1_x_); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<num_blocks, dim1_x_>>>(DATA(y), CDATA(x), k, size); \
}

#define CUDADEV_FW_X_SCALAR(name) \
void CUDADevice::name##_fw_impl(const Tensor &x, const Tensor &k, Tensor &y) { \
  const unsigned size = y.shape().volume(); \
  const unsigned g1 = GRID_SIZE(size, dim1_x_); \
  const unsigned g2 = y.shape().batch(); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<dim3(g1, g2, 1), dim1_x_>>>( \
      DATA(y), CDATA(x), CDATA(k), size, \
      x.shape().has_batch(), k.shape().has_batch()); \
}

#define CUDADEV_FW_AB(name) \
void CUDADevice::name##_fw_impl(const Tensor &a, const Tensor &b, Tensor &y) { \
  const unsigned size = y.shape().volume(); \
  const unsigned g1 = GRID_SIZE(size, dim1_x_); \
  const unsigned g2 = y.shape().batch(); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<dim3(g1, g2, 1), dim1_x_>>>( \
      DATA(y), CDATA(a), CDATA(b), size, \
      a.shape().has_batch(), b.shape().has_batch()); \
}

CUDADEV_FW_X(negate);
CUDADEV_FW_X(sqrt);
CUDADEV_FW_X(exp);
CUDADEV_FW_X(tanh);
CUDADEV_FW_X(sigmoid);
CUDADEV_FW_X(sin);
CUDADEV_FW_X(cos);
CUDADEV_FW_X(tan);

CUDADEV_FW_X_CONST(add_const);
CUDADEV_FW_X_CONST(subtract_const_r);
CUDADEV_FW_X_CONST(subtract_const_l);
CUDADEV_FW_X_CONST(multiply_const);
CUDADEV_FW_X_CONST(divide_const_r);
CUDADEV_FW_X_CONST(divide_const_l);
CUDADEV_FW_X_CONST(pstep);
CUDADEV_FW_X_CONST(prelu);

CUDADEV_FW_X_SCALAR(add_scalar);
CUDADEV_FW_X_SCALAR(subtract_scalar_r);
CUDADEV_FW_X_SCALAR(subtract_scalar_l);
CUDADEV_FW_X_SCALAR(multiply_scalar);
CUDADEV_FW_X_SCALAR(divide_scalar_r);
CUDADEV_FW_X_SCALAR(divide_scalar_l);

CUDADEV_FW_AB(add);
CUDADEV_FW_AB(subtract);
CUDADEV_FW_AB(multiply);
CUDADEV_FW_AB(divide);

#undef CUDADEV_FW_X
#undef CUDADEV_FW_X_CONST
#undef CUDADEV_FW_X_SCALAR
#undef CUDADEV_FW_AB

void CUDADevice::transpose_fw_impl(const Tensor &x, Tensor &y) {
  const unsigned d1 = x.shape()[0];
  const unsigned d2 = x.shape()[1];
  const unsigned bs = x.shape().batch();
  const unsigned g1 = GRID_SIZE(d1, dim2_x_);
  const unsigned g2 = GRID_SIZE(d2, dim2_y_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::transpose_fw_dev<<<dim3(g1, g2, bs), dim3(dim2_x_, dim2_y_, 1)>>>(
      DATA(y), CDATA(x), d1, d2);
}

void CUDADevice::matmul_fw_impl(const Tensor &a, const Tensor &b, Tensor &y) {
  const unsigned di = a.shape()[0];
  const unsigned dj = a.shape()[1];
  const unsigned dk = b.shape()[1];
  float alpha = 1.;
  float beta = 0.;
  CUDA_CALL(::hipSetDevice(dev_id_));
  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const unsigned a_skip = di * dj;
    const unsigned b_skip = b.shape().has_batch() * dj * dk;
    const unsigned y_skip = di * dk;
    const unsigned bs = a.shape().batch();
    for (unsigned n = 0; n < bs; ++n) {
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
            di, dk, dj,
            &alpha, CDATA(a) + n * a_skip, di, CDATA(b) + n * b_skip, dj,
            &beta, DATA(y) + n * y_skip, di));
    }
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
          di, dk * b.shape().batch(), dj,
          &alpha, CDATA(a), di, CDATA(b), dj,
          &beta, DATA(y), di));
  }
}

void CUDADevice::matmul_bw_impl(
    const Tensor &a, const Tensor &b, const Tensor &gy,
    Tensor &ga, Tensor &gb) {
  // ga += gy . b^T
  // gb += a^T . gy
  const unsigned di = a.shape()[0];
  const unsigned dj = a.shape()[1];
  const unsigned dk = b.shape()[1];
  float alpha = 1.;
  float beta = 1.;
  CUDA_CALL(::hipSetDevice(dev_id_));
  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const unsigned a_skip = di * dj;
    const unsigned b_skip = b.shape().has_batch() * dj * dk;
    const unsigned y_skip = di * dk;
    const unsigned bs = a.shape().batch();
    for (unsigned n = 0; n < bs; ++n) {
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
            di, dj, dk,
            &alpha, CDATA(gy) + n * y_skip, di, CDATA(b) + n * b_skip, dj,
            &beta, DATA(ga) + n * a_skip, di));
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
            dj, dk, di,
            &alpha, CDATA(a) + n * a_skip, di, CDATA(gy) + n * y_skip, di,
            &beta, DATA(gb) + n * b_skip, dj));
    }
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
          di, dj, dk * b.shape().batch(),
          &alpha, CDATA(gy), di, CDATA(b), dj,
          &beta, DATA(ga), di));
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
          dj, dk * b.shape().batch(), di,
          &alpha, CDATA(a), di, CDATA(gy), di,
          &beta, DATA(gb), dj));
  }
}

void CUDADevice::sum_fw_impl(const Tensor &x, unsigned dim, Tensor &y) {
  const unsigned n = x.shape()[dim];
  const unsigned r = y.shape().size();
  const unsigned s = y.shape().lower_volume(dim);
  unsigned block_size = dim1_x_;
  while (block_size >> 1 >= n) block_size >>= 1;
  CUDA_CALL(::hipSetDevice(dev_id_));
  switch (block_size) {
#define CASE(k) \
    case k: ::sum_fw_dev<k><<<r, k>>>(DATA(y), CDATA(x), s, n); break
    CASE(1024);
    CASE(512);
    CASE(256);
    CASE(128);
    CASE(64);
    CASE(32);
    CASE(16);
    CASE(8);
    CASE(4);
    CASE(2);
    CASE(1);
#undef CASE
  }
}

void CUDADevice::logsumexp_fw_impl(const Tensor &x, unsigned dim, Tensor &y) {
  const unsigned n = x.shape()[dim];
  const unsigned r = y.shape().size();
  const unsigned s = y.shape().lower_volume(dim);
  unsigned block_size = dim1_x_;
  while (block_size >> 1 >= n) block_size >>= 1;
  CUDA_CALL(::hipSetDevice(dev_id_));
  switch (block_size) {
#define CASE(k) \
    case k: ::logsumexp_fw_dev<k><<<r, k>>>(DATA(y), CDATA(x), s, n); break
    CASE(1024);
    CASE(512);
    CASE(256);
    CASE(128);
    CASE(64);
    CASE(32);
    CASE(16);
    CASE(8);
    CASE(4);
    CASE(2);
    CASE(1);
#undef CASE
  }
}

void CUDADevice::broadcast_fw_impl(
    const Tensor &x, unsigned dim, unsigned size, Tensor &y) {
  const unsigned skip1 = y.shape().lower_volume(dim);
  const unsigned skip2 = skip1 * size;
  const unsigned total = y.shape().size();
  const unsigned g1 = GRID_SIZE(total, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::broadcast_fw_dev<<<g1, dim1_x_>>>(DATA(y), CDATA(x), skip1, skip2, total);
}

void CUDADevice::batch_sum_fw_impl(const Tensor &x, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned g1 = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::batch_sum_fw_dev<<<g1, dim1_x_>>>(
      DATA(y), CDATA(x), size, x.shape().batch());
}

void CUDADevice::add_gradient_impl(Tensor &a, const Tensor &b) {
  const unsigned nx = a.shape().size();
  const unsigned ny = b.shape().size();
  const unsigned g1 = GRID_SIZE(std::max(nx, ny), dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::add_grad_dev<<<g1, dim1_x_>>>(DATA(a), CDATA(b), nx, ny);
}

void CUDADevice::add_gradient_offset_impl(
    Tensor &a, const Tensor &b, unsigned dim, unsigned offset) {
  const Shape &sa = a.shape();
  const Shape &sb = b.shape();
  const unsigned base = sa.lower_volume(dim);
  unsigned repeat = 1;
  for (unsigned i = dim + 1; i < sa.depth(); ++i) repeat *= sa[i];
  const unsigned ox = base * offset;
  const unsigned wx = base * sa[dim];
  const unsigned wy = base * sb[dim];
  const unsigned nx = repeat * sa.batch();
  const unsigned ny = repeat * sb.batch();
  const unsigned g1 = GRID_SIZE(wy * std::max(nx, ny), dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::add_grad_ofs_dev<<<g1, dim1_x_>>>(DATA(a) + ox, CDATA(b), wx, wy, nx, ny);
}

void CUDADevice::add_gradient_sparse_impl(
    Tensor &a, const Tensor &b,
    unsigned dim, const std::vector<unsigned>& ids) {
  const Shape &sa = a.shape();
  const Shape &sb = b.shape();
  const unsigned size = sb.volume();
  const unsigned base = sb.lower_volume(dim);
  const unsigned repeat = size / base;
  const unsigned wx = base * sa[dim];
  const unsigned g1 = GRID_SIZE(size, dim1_x_);
  const unsigned bs = sb.batch();
  const unsigned skip_a = (sa.has_batch()) * sa.volume();
  const unsigned skip_i = ids.size() > 1;
  float *dest = DATA(a);
  const float *src = CDATA(b);

  CUDA_CALL(::hipSetDevice(dev_id_));
  for (unsigned batch = 0; batch < bs; ++batch) {
    ::add_grad_sparse_dev<<<g1, dim1_x_>>>(
        dest + batch * skip_a + base * ids[batch * skip_i],
        src + batch * size,
        wx, base, repeat);
  }
}

}  // namespace primitiv
