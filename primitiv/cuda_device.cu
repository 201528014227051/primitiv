#include "hip/hip_runtime.h"
#include <config.h>

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <random>
#include <primitiv/cuda_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/error.h>

using std::cerr;
using std::endl;

namespace {

/*
 * CUDA kernels
 */

#define IDX (threadIdx.x + blockIdx.x * blockDim.x)

__global__ void set_const_dev(float k, unsigned size, float *py) {
  const unsigned i = IDX;
  if (i < size) py[i] = k;
}

__global__ void rand_bernoulli_dev(float p, float size, float *py) {
  const unsigned i = IDX;
  if (i < size) py[i] = (float)(py[i] <= p);
}

__global__ void rand_affine_dev(
    float shift, float scale, unsigned size, float *py) {
  const unsigned i = IDX;
  if (i < size) py[i] = py[i] * scale + shift;
}

__global__ void slice_fw_dev(
    const float *px, unsigned span, unsigned skip, unsigned size, float *py) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[(i / span) * skip + (i % span)];
}

__global__ void concat_fw_dev(
    const float *px, unsigned span, unsigned skip, unsigned x_size,
    unsigned y_size, float *py) {
  const unsigned i = IDX;
  if (i < y_size) py[(i / span) * skip + (i % span)] = px[i % x_size];
}

#define CUDADEV_KERNEL_FW_X(name, op) \
__global__ void name##_fw_dev(const float *px, unsigned size, float *py) { \
  const unsigned i = IDX; \
  if (i < size) py[i] = (op); \
}

#define CUDADEV_KERNEL_BW_X(name, op) \
__global__ void name##_bw_dev( \
    const float *px, const float *py, const float *pgy, unsigned size, \
    float *pgx) { \
  static_cast<void>(px); \
  static_cast<void>(py); \
  const unsigned i = IDX; \
  if (i < size) pgx[i] += (op); \
}

#define CUDADEV_KERNEL_FW_X_CONST(name, op) \
__global__ void name##_fw_dev( \
    const float *px, float k, unsigned size, float *py) { \
  const unsigned i = IDX; \
  if (i < size) py[i] = (op); \
}

#define CUDADEV_KERNEL_FW_X_SCALAR_R(name, op) \
__global__ void name##_fw_dev( \
    const float *px, const float *pk, unsigned size, unsigned mbx, \
    unsigned mbk, float *py) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(px[i + mbx * shift], pk[mbk * blockIdx.y]); \
}

#define CUDADEV_KERNEL_FW_X_SCALAR_L(name, op) \
__global__ void name##_fw_dev( \
    const float *px, const float *pk, unsigned size, unsigned mbx, \
    unsigned mbk, float *py) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(pk[mbk * blockIdx.y], px[i + mbx * shift]); \
}

#define CUDADEV_KERNEL_FW_AB(name, op) \
__global__ void name##_fw_dev( \
    const float *pa, const float *pb, unsigned size, unsigned mba, \
    unsigned mbb, float *py) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(pa[i + mba * shift], pb[i + mbb * shift]); \
}

CUDADEV_KERNEL_FW_X(negate, -px[i]);
CUDADEV_KERNEL_FW_X(sqrt, ::__fsqrt_rn(px[i]));
CUDADEV_KERNEL_FW_X(exp, ::expf(px[i]));
CUDADEV_KERNEL_FW_X(tanh, ::tanhf(px[i]));
CUDADEV_KERNEL_FW_X(sigmoid, .5f + .5f * ::tanhf(.5f * px[i]));
CUDADEV_KERNEL_FW_X(sin, ::sinf(px[i]));
CUDADEV_KERNEL_FW_X(cos, ::cosf(px[i]));
CUDADEV_KERNEL_FW_X(tan, ::tanf(px[i]));

CUDADEV_KERNEL_BW_X(negate, -pgy[i]);
CUDADEV_KERNEL_BW_X(sqrt, .5f * pgy[i] / py[i]);
CUDADEV_KERNEL_BW_X(exp, py[i] * pgy[i]);
CUDADEV_KERNEL_BW_X(tanh, (1.f - py[i] * py[i]) * pgy[i]);
CUDADEV_KERNEL_BW_X(sigmoid, py[i] * (1.f - py[i]) * pgy[i]);
CUDADEV_KERNEL_BW_X(sin, ::cosf(px[i]) * pgy[i]);
CUDADEV_KERNEL_BW_X(cos, -::sinf(px[i]) * pgy[i]);
CUDADEV_KERNEL_BW_X(tan, (1.f + py[i] * py[i]) * pgy[i]);

CUDADEV_KERNEL_FW_X_CONST(add_const, px[i] + k);
CUDADEV_KERNEL_FW_X_CONST(subtract_const_r, px[i] - k);
CUDADEV_KERNEL_FW_X_CONST(subtract_const_l, k - px[i]);
CUDADEV_KERNEL_FW_X_CONST(multiply_const, px[i] * k);
CUDADEV_KERNEL_FW_X_CONST(divide_const_r, px[i] / k);
CUDADEV_KERNEL_FW_X_CONST(divide_const_l, k / px[i]);
CUDADEV_KERNEL_FW_X_CONST(pstep, (px[i] > .0f) + k * (px[i] <= .0f));
CUDADEV_KERNEL_FW_X_CONST(prelu, px[i] * ((px[i] > .0f) + k * (px[i] <= .0f)));

CUDADEV_KERNEL_FW_X_SCALAR_R(add_scalar, ::__fadd_rn);
CUDADEV_KERNEL_FW_X_SCALAR_R(subtract_scalar_r, ::__fsub_rn);
CUDADEV_KERNEL_FW_X_SCALAR_L(subtract_scalar_l, ::__fsub_rn);
CUDADEV_KERNEL_FW_X_SCALAR_R(multiply_scalar, ::__fmul_rn);
CUDADEV_KERNEL_FW_X_SCALAR_R(divide_scalar_r, ::__fdiv_rn);
CUDADEV_KERNEL_FW_X_SCALAR_L(divide_scalar_l, ::__fdiv_rn);

CUDADEV_KERNEL_FW_AB(add, ::__fadd_rn);
CUDADEV_KERNEL_FW_AB(subtract, ::__fsub_rn);
CUDADEV_KERNEL_FW_AB(multiply, ::__fmul_rn);
CUDADEV_KERNEL_FW_AB(divide, ::__fdiv_rn);

#undef CUDADEV_KERNEL_FW_X
#undef CUDADEV_KERNEL_FW_X_CONST
#undef CUDADEV_KERNEL_FW_X_SCALAR_R
#undef CUDADEV_KERNEL_FW_X_SCALAR_L

__global__ void transpose_fw_dev(
    const float *px, unsigned rows, unsigned cols, float *py) {
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned ofs = blockIdx.z * rows * cols;
  if (i < rows && j < cols) {
    py[ofs + j + i * cols] = px[ofs + i + j * rows];
  }
}

template<unsigned BLOCK_SIZE>
__global__ void sum_fw_dev(
    const float *px, unsigned skip, unsigned n, float *py) {
  __shared__ float temp[BLOCK_SIZE];
  const unsigned bid = blockIdx.x;
  const unsigned tid = threadIdx.x;
  px += bid % skip + (bid / skip) * skip * n;
  temp[tid] = 0;
  for (unsigned i = tid; i < n; i += BLOCK_SIZE) temp[tid] += px[i * skip];
  __syncthreads();
#define REDUCE(k) \
  if (BLOCK_SIZE >= k << 1) { \
    if (tid < k) temp[tid] += temp[tid + k]; \
    __syncthreads(); \
  }
  REDUCE(512)
  REDUCE(256)
  REDUCE(128)
  REDUCE(64)
  REDUCE(32)
  REDUCE(16)
  REDUCE(8)
  REDUCE(4)
  REDUCE(2)
  REDUCE(1)
#undef REDUCE
  if (tid == 0) py[bid] = temp[0];
}

__device__ float logsumexp2_fw_dev(float a, float b) {
  return a > b
    ? a + ::log(1.f + ::exp(b - a))
    : b + ::log(1.f + ::exp(a - b));
}

template<unsigned BLOCK_SIZE>
__global__ void logsumexp_fw_dev(
    const float *px, unsigned skip, unsigned n, float *py) {
  __shared__ float temp[BLOCK_SIZE];
  const unsigned bid = blockIdx.x;
  const unsigned tid = threadIdx.x;
  px += bid % skip + (bid / skip) * skip * n;
  temp[tid] = -1e38;  // NOTE(odashi): Near the minimum of the float.
  for (unsigned i = tid; i < n; i += BLOCK_SIZE) {
    temp[tid] = ::logsumexp2_fw_dev(temp[tid], px[i * skip]);
  }
  __syncthreads();
#define REDUCE(k) \
  if (BLOCK_SIZE >= k << 1) { \
    if (tid < k) temp[tid] = ::logsumexp2_fw_dev(temp[tid], temp[tid + k]); \
    __syncthreads(); \
  }
  REDUCE(512)
  REDUCE(256)
  REDUCE(128)
  REDUCE(64)
  REDUCE(32)
  REDUCE(16)
  REDUCE(8)
  REDUCE(4)
  REDUCE(2)
  REDUCE(1)
#undef REDUCE
  if (tid == 0) py[bid] = temp[0];
}

__global__ void broadcast_fw_dev(
    const float *px, unsigned skip1, unsigned skip2, unsigned size, float *py) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[i % skip1 + (i / skip2) * skip1];
}

__global__ void batch_sum_fw_dev(
    const float *px, unsigned size, unsigned batch, float *py) {
  const unsigned i = IDX;
  if (i < size) {
    float temp = .0f;
    px += i;
    for (unsigned j = 0; j < batch; ++j, px += size) {
      temp += *px;
    }
    py[i] = temp;
  }
}

__global__ void add_grad_dev(
    const float *pgy, unsigned nx, unsigned ny, float *pgx) {
  const unsigned i = IDX;
  if (i < ::max(nx, ny)) ::atomicAdd(pgx + i % nx, pgy[i % ny]);
}

__global__ void add_grad_ofs_dev(
    const float *pgy, unsigned wx, unsigned wy, unsigned nx, unsigned ny,
    float *pgx) {
  const unsigned i = IDX;
  if (i < wy * ::max(nx, ny)) {
    ::atomicAdd(
        pgx + ((i / wy) * wx + (i % wy)) % (wx * nx),
        pgy[i % (wy * ny)]);
  }
}

__global__ void add_grad_sparse_dev(
    const float *pgy, unsigned wx, unsigned wy, unsigned repeat, float *pgx) {
  const unsigned i = IDX;
  if (i < wy * repeat) {
    ::atomicAdd(pgx + (i / wy) * wx + (i % wy), pgy[i]);
  }
}

#undef IDX

}  // namespace

namespace {

// Minimum requirements of the compute capability.
static const int MIN_CC_MAJOR = 3;
static const int MIN_CC_MINOR = 0;

}

namespace primitiv {

unsigned CUDADevice::num_devices() {
  int ret;
  CUDA_CALL(::hipGetDeviceCount(&ret));
  return ret;
}

void CUDADevice::initialize() {
  // Retrieves device properties.
  ::hipDeviceProp_t prop;
  CUDA_CALL(::hipGetDeviceProperties(&prop, dev_id_));

  // Dump device properties.
  cerr << "Selected CUDA Device " << dev_id_ << ':' << endl;
  cerr << "  Name ................. " << prop.name << endl;
  cerr << "  Global Memory ........ " << prop.totalGlobalMem << endl;
  cerr << "  Shared Memory ........ " << prop.sharedMemPerBlock << endl;
  cerr << "  Threads/block ........ " << prop.maxThreadsPerBlock << endl;
  cerr << "  Threads dim .......... " << prop.maxThreadsDim[0] << ','
                                      << prop.maxThreadsDim[1] << ','
                                      << prop.maxThreadsDim[2] << endl;
  cerr << "  Grid size ............ " << prop.maxGridSize[0] << ','
                                      << prop.maxGridSize[1] << ','
                                      << prop.maxGridSize[2] << endl;
  cerr << "  Compute Capability ... " << prop.major << '.'
                                      << prop.minor << endl;

  // Check compute capability requirements.
  if (prop.major < ::MIN_CC_MAJOR ||
      (prop.major == ::MIN_CC_MAJOR && prop.minor < ::MIN_CC_MINOR)) {
    THROW_ERROR(
        "CUDA Device " << dev_id_ << " does not satisfy the "
        "minimum requirement of the compute capability: "
        << prop.major << '.' << prop.minor << " < "
        << ::MIN_CC_MAJOR << '.' << ::MIN_CC_MINOR);
  }

  // Calculates size of dims to be used in CUDA kernels.
  dim1_x_ = 1;
  while (dim1_x_ < 1024 &&
      dim1_x_ < static_cast<unsigned>(prop.maxThreadsPerBlock)) {
    dim1_x_ <<= 1;
  }
  dim2_y_ = dim1_x_;
  dim2_x_ = 1;
  while (dim2_x_ < dim2_y_) {
    dim2_x_ <<= 1;
    dim2_y_ >>= 1;
  }
  cerr << "Block configuration:" << endl;
  cerr << "  1 dim .... " << dim1_x_ << " threads" << endl;
  cerr << "  2 dims ... " << dim2_x_ << "x" << dim2_y_ << " threads" << endl;

  // Initializes additional libraries
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUBLAS_CALL(::hipblasCreate(&cublas_));
  CURAND_CALL(::hiprandCreateGenerator(&curand_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(::hiprandSetPseudoRandomGeneratorSeed(curand_, rng_seed_));
}

CUDADevice::CUDADevice(unsigned device_id)
: dev_id_(device_id)
, rng_seed_(std::random_device()())
, pool_(device_id) {
  initialize();
}

CUDADevice::CUDADevice(unsigned device_id, unsigned rng_seed)
: dev_id_(device_id)
, rng_seed_(rng_seed)
, pool_(device_id) {
  initialize();
}

CUDADevice::~CUDADevice() {
  // Finalizes additional libraries
  CUBLAS_CALL(::hipblasDestroy(cublas_));
  CURAND_CALL(::hiprandDestroyGenerator(curand_));
}

std::shared_ptr<void> CUDADevice::new_handle(const Shape &shape) {
  return pool_.allocate(sizeof(float) * shape.size());
}

#define GRID_SIZE(x, threads) (((x) + (threads) - 1) / (threads))
#define DATA(x) static_cast<float *>((x).data())
#define CDATA(x) static_cast<const float *>((x).data())

std::vector<float> CUDADevice::tensor_to_vector_impl(const Tensor &x) {
  const unsigned size = x.shape().size();
  std::vector<float> ret(size);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        &ret[0], x.data(), sizeof(float) * size, hipMemcpyDeviceToHost));
  return ret;
}

void CUDADevice::reset_tensor_impl(float k, Tensor &x) {
  const unsigned size = x.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_const_dev<<<num_blocks, dim1_x_>>>(k, size, DATA(x));
}

void CUDADevice::reset_tensor_by_array_impl(const float values[], Tensor &x) {
  const unsigned size = x.shape().size();
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        x.data(), values, sizeof(float) * size, hipMemcpyHostToDevice));
}

void CUDADevice::copy_tensor_impl(const Tensor &x, Tensor &y) {
  switch (x.device()->type()) {
    case Device::DEVICE_TYPE_CPU:
      reset_tensor_by_array(CDATA(x), y);
      break;
    case Device::DEVICE_TYPE_CUDA:
      CUDA_CALL(::hipSetDevice(dev_id_));
      CUDA_CALL(::hipMemcpyAsync(
            DATA(y), CDATA(x),
            sizeof(float) * x.shape().size(),
            hipMemcpyDeviceToDevice, 0));
      break;
    default:
      reset_tensor_by_vector(x.to_vector(), y);
  }
}

void CUDADevice::random_bernoulli_impl(float p, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(y), size));
  ::rand_bernoulli_dev<<<num_blocks, dim1_x_>>>(p, size, DATA(y));
}

void CUDADevice::random_uniform_impl(float lower, float upper, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  const float scale = upper - lower;
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(y), size));
  ::rand_affine_dev<<<num_blocks, dim1_x_>>>(lower, scale, size, DATA(y));
}

void CUDADevice::random_normal_impl(float mean, float sd, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateNormal(
        curand_, DATA(y), y.shape().size(), mean, sd));
}

void CUDADevice::random_log_normal_impl(float mean, float sd, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateLogNormal(
        curand_, DATA(y), y.shape().size(), mean, sd));
}

void CUDADevice::pick_fw_impl(
    const Tensor &x, unsigned dim,
    const std::vector<unsigned> &ids, Tensor &y) {
  const unsigned base = y.shape().lower_volume(dim);
  const unsigned skip = base * x.shape()[dim];
  const unsigned size = y.shape().volume();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  const unsigned skip_x = x.shape().has_batch() * x.shape().volume();
  const unsigned skip_i = ids.size() > 1;
  const unsigned bs = y.shape().batch();
  CUDA_CALL(::hipSetDevice(dev_id_));
  for (unsigned b = 0; b < bs; ++b) {
    ::slice_fw_dev<<<num_blocks, dim1_x_>>>(
        CDATA(x) + b * skip_x + base * ids[b * skip_i],
        base, skip, size, DATA(y) + b * size);
  }
}

void CUDADevice::slice_fw_impl(
    const Tensor &x, unsigned dim, unsigned offset, Tensor &y) {
  const unsigned base = y.shape().lower_volume(dim);
  const unsigned span = base * y.shape()[dim];
  const unsigned skip = base * x.shape()[dim];
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::slice_fw_dev<<<num_blocks, dim1_x_>>>(
      CDATA(x) + base * offset, span, skip, size, DATA(y));
}

void CUDADevice::concat_fw_impl(
    const std::vector<const Tensor *> &xs, unsigned dim, Tensor &y) {
  const unsigned new_bs = y.shape().batch();
  const unsigned base = y.shape().lower_volume(dim);
  const unsigned skip = base * y.shape()[dim];
  unsigned repeat = y.shape().volume() / skip;
  CUDA_CALL(::hipSetDevice(dev_id_));
  unsigned offset = 0;
  for (const Tensor *x : xs) {
    const unsigned span = base * x->shape()[dim];
    const unsigned x_size = span * repeat * x->shape().batch();
    const unsigned y_size = span * repeat * new_bs;
    const unsigned num_blocks = GRID_SIZE(y_size, dim1_x_);
    ::concat_fw_dev<<<num_blocks, dim1_x_>>>(
       CDATA(*x), span, skip, x_size, y_size, DATA(y) + offset);
    offset += span;
  }
}

#define CUDADEV_FW_X(name) \
void CUDADevice::name##_fw_impl(const Tensor &x, Tensor &y) { \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<num_blocks, dim1_x_>>>(CDATA(x), size, DATA(y)); \
}

#define CUDADEV_BW_X(name) \
void CUDADevice::name##_bw_impl( \
    const Tensor &x, const Tensor &y, const Tensor &gy, Tensor &gx) { \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_bw_dev<<<num_blocks, dim1_x_>>>( \
      CDATA(x), CDATA(y), CDATA(gy), size, DATA(gx)); \
}

#define CUDADEV_FW_X_CONST(name) \
void CUDADevice::name##_fw_impl(const Tensor &x, float k, Tensor &y) { \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size,dim1_x_); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<num_blocks, dim1_x_>>>(CDATA(x), k, size, DATA(y)); \
}

#define CUDADEV_FW_X_SCALAR(name) \
void CUDADevice::name##_fw_impl(const Tensor &x, const Tensor &k, Tensor &y) { \
  const unsigned size = y.shape().volume(); \
  const unsigned g1 = GRID_SIZE(size, dim1_x_); \
  const unsigned g2 = y.shape().batch(); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<dim3(g1, g2, 1), dim1_x_>>>( \
      CDATA(x), CDATA(k), size, \
      x.shape().has_batch(), k.shape().has_batch(), DATA(y)); \
}

#define CUDADEV_FW_AB(name) \
void CUDADevice::name##_fw_impl(const Tensor &a, const Tensor &b, Tensor &y) { \
  const unsigned size = y.shape().volume(); \
  const unsigned g1 = GRID_SIZE(size, dim1_x_); \
  const unsigned g2 = y.shape().batch(); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<dim3(g1, g2, 1), dim1_x_>>>( \
      CDATA(a), CDATA(b), size, \
      a.shape().has_batch(), b.shape().has_batch(), DATA(y)); \
}

CUDADEV_FW_X(negate);
CUDADEV_FW_X(sqrt);
CUDADEV_FW_X(exp);
CUDADEV_FW_X(tanh);
CUDADEV_FW_X(sigmoid);
CUDADEV_FW_X(sin);
CUDADEV_FW_X(cos);
CUDADEV_FW_X(tan);

CUDADEV_BW_X(negate);
CUDADEV_BW_X(sqrt);
CUDADEV_BW_X(exp);
CUDADEV_BW_X(tanh);
CUDADEV_BW_X(sigmoid);
CUDADEV_BW_X(sin);
CUDADEV_BW_X(cos);
CUDADEV_BW_X(tan);

CUDADEV_FW_X_CONST(add_const);
CUDADEV_FW_X_CONST(subtract_const_r);
CUDADEV_FW_X_CONST(subtract_const_l);
CUDADEV_FW_X_CONST(multiply_const);
CUDADEV_FW_X_CONST(divide_const_r);
CUDADEV_FW_X_CONST(divide_const_l);
CUDADEV_FW_X_CONST(pstep);
CUDADEV_FW_X_CONST(prelu);

CUDADEV_FW_X_SCALAR(add_scalar);
CUDADEV_FW_X_SCALAR(subtract_scalar_r);
CUDADEV_FW_X_SCALAR(subtract_scalar_l);
CUDADEV_FW_X_SCALAR(multiply_scalar);
CUDADEV_FW_X_SCALAR(divide_scalar_r);
CUDADEV_FW_X_SCALAR(divide_scalar_l);

CUDADEV_FW_AB(add);
CUDADEV_FW_AB(subtract);
CUDADEV_FW_AB(multiply);
CUDADEV_FW_AB(divide);

#undef CUDADEV_FW_X
#undef CUDADEV_FW_X_CONST
#undef CUDADEV_FW_X_SCALAR
#undef CUDADEV_FW_AB

void CUDADevice::transpose_fw_impl(const Tensor &x, Tensor &y) {
  const unsigned d1 = x.shape()[0];
  const unsigned d2 = x.shape()[1];
  const unsigned bs = x.shape().batch();
  const unsigned g1 = GRID_SIZE(d1, dim2_x_);
  const unsigned g2 = GRID_SIZE(d2, dim2_y_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::transpose_fw_dev<<<dim3(g1, g2, bs), dim3(dim2_x_, dim2_y_, 1)>>>(
      CDATA(x), d1, d2, DATA(y));
}

void CUDADevice::matmul_fw_impl(const Tensor &a, const Tensor &b, Tensor &y) {
  const unsigned di = a.shape()[0];
  const unsigned dj = a.shape()[1];
  const unsigned dk = b.shape()[1];
  float alpha = 1.;
  float beta = 0.;
  CUDA_CALL(::hipSetDevice(dev_id_));
  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const unsigned a_skip = di * dj;
    const unsigned b_skip = b.shape().has_batch() * dj * dk;
    const unsigned y_skip = di * dk;
    const unsigned bs = a.shape().batch();
    for (unsigned n = 0; n < bs; ++n) {
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
            di, dk, dj,
            &alpha, CDATA(a) + n * a_skip, di, CDATA(b) + n * b_skip, dj,
            &beta, DATA(y) + n * y_skip, di));
    }
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
          di, dk * b.shape().batch(), dj,
          &alpha, CDATA(a), di, CDATA(b), dj,
          &beta, DATA(y), di));
  }
}

void CUDADevice::matmul_bw_impl(
    const Tensor &a, const Tensor &b, const Tensor &gy,
    Tensor &ga, Tensor &gb) {
  // ga += gy . b^T
  // gb += a^T . gy
  const unsigned di = a.shape()[0];
  const unsigned dj = a.shape()[1];
  const unsigned dk = b.shape()[1];
  float alpha = 1.;
  float beta = 1.;
  CUDA_CALL(::hipSetDevice(dev_id_));
  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const unsigned a_skip = di * dj;
    const unsigned b_skip = b.shape().has_batch() * dj * dk;
    const unsigned y_skip = di * dk;
    const unsigned bs = a.shape().batch();
    for (unsigned n = 0; n < bs; ++n) {
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
            di, dj, dk,
            &alpha, CDATA(gy) + n * y_skip, di, CDATA(b) + n * b_skip, dj,
            &beta, DATA(ga) + n * a_skip, di));
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
            dj, dk, di,
            &alpha, CDATA(a) + n * a_skip, di, CDATA(gy) + n * y_skip, di,
            &beta, DATA(gb) + n * b_skip, dj));
    }
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
          di, dj, dk * b.shape().batch(),
          &alpha, CDATA(gy), di, CDATA(b), dj,
          &beta, DATA(ga), di));
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
          dj, dk * b.shape().batch(), di,
          &alpha, CDATA(a), di, CDATA(gy), di,
          &beta, DATA(gb), dj));
  }
}

void CUDADevice::sum_fw_impl(const Tensor &x, unsigned dim, Tensor &y) {
  const unsigned n = x.shape()[dim];
  const unsigned r = y.shape().size();
  const unsigned s = y.shape().lower_volume(dim);
  unsigned block_size = dim1_x_;
  while (block_size >> 1 >= n) block_size >>= 1;
  CUDA_CALL(::hipSetDevice(dev_id_));
  switch (block_size) {
#define CASE(k) \
    case k: ::sum_fw_dev<k><<<r, k>>>(CDATA(x), s, n, DATA(y)); break
    CASE(1024);
    CASE(512);
    CASE(256);
    CASE(128);
    CASE(64);
    CASE(32);
    CASE(16);
    CASE(8);
    CASE(4);
    CASE(2);
    CASE(1);
#undef CASE
  }
}

void CUDADevice::logsumexp_fw_impl(const Tensor &x, unsigned dim, Tensor &y) {
  const unsigned n = x.shape()[dim];
  const unsigned r = y.shape().size();
  const unsigned s = y.shape().lower_volume(dim);
  unsigned block_size = dim1_x_;
  while (block_size >> 1 >= n) block_size >>= 1;
  CUDA_CALL(::hipSetDevice(dev_id_));
  switch (block_size) {
#define CASE(k) \
    case k: ::logsumexp_fw_dev<k><<<r, k>>>(CDATA(x), s, n, DATA(y)); break
    CASE(1024);
    CASE(512);
    CASE(256);
    CASE(128);
    CASE(64);
    CASE(32);
    CASE(16);
    CASE(8);
    CASE(4);
    CASE(2);
    CASE(1);
#undef CASE
  }
}

void CUDADevice::broadcast_fw_impl(
    const Tensor &x, unsigned dim, unsigned size, Tensor &y) {
  const unsigned skip1 = y.shape().lower_volume(dim);
  const unsigned skip2 = skip1 * size;
  const unsigned total = y.shape().size();
  const unsigned g1 = GRID_SIZE(total, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::broadcast_fw_dev<<<g1, dim1_x_>>>(CDATA(x), skip1, skip2, total, DATA(y));
}

void CUDADevice::batch_sum_fw_impl(const Tensor &x, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned g1 = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::batch_sum_fw_dev<<<g1, dim1_x_>>>(
      CDATA(x), size, x.shape().batch(), DATA(y));
}

void CUDADevice::add_gradient_impl(const Tensor &gy, Tensor &gx) {
  const unsigned nx = gx.shape().size();
  const unsigned ny = gy.shape().size();
  const unsigned g1 = GRID_SIZE(std::max(nx, ny), dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::add_grad_dev<<<g1, dim1_x_>>>(CDATA(gy), nx, ny, DATA(gx));
}

void CUDADevice::add_gradient_offset_impl(
    const Tensor &gy, unsigned dim, unsigned offset, Tensor &gx) {
  const Shape &sx = gx.shape();
  const Shape &sy = gy.shape();
  const unsigned base = sx.lower_volume(dim);
  const unsigned ox = base * offset;
  const unsigned wx = base * sx[dim];
  const unsigned wy = base * sy[dim];
  const unsigned repeat = sx.volume() / wx;
  const unsigned nx = repeat * sx.batch();
  const unsigned ny = repeat * sy.batch();
  const unsigned g1 = GRID_SIZE(wy * std::max(nx, ny), dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::add_grad_ofs_dev<<<g1, dim1_x_>>>(CDATA(gy), wx, wy, nx, ny, DATA(gx) + ox);
}

void CUDADevice::add_gradient_sparse_impl(
    const Tensor &gy, unsigned dim, const std::vector<unsigned>& ids,
    Tensor &gx) {
  const Shape &sx = gx.shape();
  const Shape &sy = gy.shape();
  const unsigned size = sy.volume();
  const unsigned base = sy.lower_volume(dim);
  const unsigned repeat = size / base;
  const unsigned wx = base * sx[dim];
  const unsigned g1 = GRID_SIZE(size, dim1_x_);
  const unsigned bs = sy.batch();
  const unsigned skip_a = (sx.has_batch()) * sx.volume();
  const unsigned skip_i = ids.size() > 1;
  float *dest = DATA(gx);
  const float *src = CDATA(gy);

  CUDA_CALL(::hipSetDevice(dev_id_));
  for (unsigned batch = 0; batch < bs; ++batch) {
    ::add_grad_sparse_dev<<<g1, dim1_x_>>>(
        src + batch * size,
        wx, base, repeat,
        dest + batch * skip_a + base * ids[batch * skip_i]);
  }
}

}  // namespace primitiv
