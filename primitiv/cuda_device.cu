#include "hip/hip_runtime.h"
#include <config.h>

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <random>
#include <primitiv/cuda_device.h>
#include <primitiv/cuda_utils.h>
#include <primitiv/error.h>

using std::cerr;
using std::endl;

namespace {

/*
 * CUDA kernels
 */

#define IDX (threadIdx.x + blockIdx.x * blockDim.x)

__global__ void set_const_dev(float *py, float k, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = k;
}

__global__ void rand_bernoulli_dev(float *px, float p, float size) {
  const unsigned i = IDX;
  if (i < size) px[i] = (float)(px[i] <= p);
}

__global__ void rand_affine_dev(
    float *px, float shift, float scale, unsigned size) {
  const unsigned i = IDX;
  if (i < size) px[i] = px[i] * scale + shift;
}

__global__ void slice_fw_dev(
    float *py, const float *px, unsigned span, unsigned skip, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[(i / span) * skip + (i % span)];
}

__global__ void concat_fw_dev(
    float *py, const float *px,
    unsigned span, unsigned skip, unsigned x_size, unsigned y_size) {
  const unsigned i = IDX;
  if (i < y_size) py[(i / span) * skip + (i % span)] = px[i % x_size];
}

#define CUDA_KERNEL_X(name, op) \
__global__ void name##_fw_dev(float *py, const float *px, unsigned size) { \
  const unsigned i = IDX; \
  if (i < size) py[i] = (op); \
}

#define CUDA_KERNEL_X_CONST(name, op) \
__global__ void name##_fw_dev(float *py, const float *px, float k, unsigned size) { \
  const unsigned i = IDX; \
  if (i < size) py[i] = (op); \
}

#define CUDA_KERNEL_X_SCALAR_R(name, op) \
__global__ void name##_fw_dev( \
    float *py, const float *px, const float *pk, \
    unsigned size, unsigned mbx, unsigned mbk) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(px[i + mbx * shift], pk[mbk * blockIdx.y]); \
}

#define CUDA_KERNEL_X_SCALAR_L(name, op) \
__global__ void name##_fw_dev( \
    float *py, const float *px, const float *pk, \
    unsigned size, unsigned mbx, unsigned mbk) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(pk[mbk * blockIdx.y], px[i + mbx * shift]); \
}

#define CUDA_KERNEL_AB(name, op) \
__global__ void name##_fw_dev( \
    float *py, const float *pa, const float *pb, \
    unsigned size, unsigned mba, unsigned mbb) { \
  const unsigned i = IDX; \
  const unsigned shift = blockIdx.y * size; \
  if (i < size) py[i + shift] = op(pa[i + mba * shift], pb[i + mbb * shift]); \
}

CUDA_KERNEL_X(negate, -px[i]);
CUDA_KERNEL_X(sqrt, ::__fsqrt_rn(px[i]));
CUDA_KERNEL_X(exp, ::expf(px[i]));
CUDA_KERNEL_X(tanh, ::tanhf(px[i]));
CUDA_KERNEL_X(sigmoid, .5f + .5f * ::tanhf(.5f * px[i]));
CUDA_KERNEL_X(sin, ::sinf(px[i]));
CUDA_KERNEL_X(cos, ::cosf(px[i]));
CUDA_KERNEL_X(tan, ::tanf(px[i]));

CUDA_KERNEL_X_CONST(add_const, px[i] + k);
CUDA_KERNEL_X_CONST(subtract_const_r, px[i] - k);
CUDA_KERNEL_X_CONST(subtract_const_l, k - px[i]);
CUDA_KERNEL_X_CONST(multiply_const, px[i] * k);
CUDA_KERNEL_X_CONST(divide_const_r, px[i] / k);
CUDA_KERNEL_X_CONST(divide_const_l, k / px[i]);
CUDA_KERNEL_X_CONST(pstep, (px[i] > .0f) + k * (px[i] <= .0f));
CUDA_KERNEL_X_CONST(prelu, px[i] * ((px[i] > .0f) + k * (px[i] <= .0f)));

CUDA_KERNEL_X_SCALAR_R(add_scalar, ::__fadd_rn);
CUDA_KERNEL_X_SCALAR_R(subtract_scalar_r, ::__fsub_rn);
CUDA_KERNEL_X_SCALAR_L(subtract_scalar_l, ::__fsub_rn);
CUDA_KERNEL_X_SCALAR_R(multiply_scalar, ::__fmul_rn);
CUDA_KERNEL_X_SCALAR_R(divide_scalar_r, ::__fdiv_rn);
CUDA_KERNEL_X_SCALAR_L(divide_scalar_l, ::__fdiv_rn);

CUDA_KERNEL_AB(add, ::__fadd_rn);
CUDA_KERNEL_AB(subtract, ::__fsub_rn);
CUDA_KERNEL_AB(multiply, ::__fmul_rn);
CUDA_KERNEL_AB(divide, ::__fdiv_rn);

#undef CUDA_KERNEL_X
#undef CUDA_KERNEL_X_CONST
#undef CUDA_KERNEL_X_SCALAR_R
#undef CUDA_KERNEL_X_SCALAR_L

__global__ void transpose_fw_dev(
    float *py, const float *px, unsigned rows, unsigned cols) {
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned ofs = blockIdx.z * rows * cols;
  if (i < rows && j < cols) {
    py[ofs + j + i * cols] = px[ofs + i + j * rows];
  }
}

template<unsigned BLOCK_SIZE>
__global__ void sum_fw_dev(float *py, const float *px, unsigned skip, unsigned n) {
  __shared__ float temp[BLOCK_SIZE];
  const unsigned bid = blockIdx.x;
  const unsigned tid = threadIdx.x;
  px += bid % skip + (bid / skip) * skip * n;
  temp[tid] = 0;
  for (unsigned i = tid; i < n; i += BLOCK_SIZE) temp[tid] += px[i * skip];
  __syncthreads();
#define REDUCE(k) \
  if (BLOCK_SIZE >= k << 1) { \
    if (tid < k) temp[tid] += temp[tid + k]; \
    __syncthreads(); \
  }
  REDUCE(512)
  REDUCE(256)
  REDUCE(128)
  REDUCE(64)
  REDUCE(32)
  REDUCE(16)
  REDUCE(8)
  REDUCE(4)
  REDUCE(2)
  REDUCE(1)
#undef REDUCE
  if (tid == 0) py[bid] = temp[0];
}

__device__ float logsumexp2_fw_dev(float a, float b) {
  return a > b
    ? a + ::log(1.f + ::exp(b - a))
    : b + ::log(1.f + ::exp(a - b));
}

template<unsigned BLOCK_SIZE>
__global__ void logsumexp_fw_dev(
    float *py, const float *px, unsigned skip, unsigned n) {
  __shared__ float temp[BLOCK_SIZE];
  const unsigned bid = blockIdx.x;
  const unsigned tid = threadIdx.x;
  px += bid % skip + (bid / skip) * skip * n;
  temp[tid] = -1e38;  // NOTE(odashi): Near the minimum of the float.
  for (unsigned i = tid; i < n; i += BLOCK_SIZE) {
    temp[tid] = ::logsumexp2_fw_dev(temp[tid], px[i * skip]);
  }
  __syncthreads();
#define REDUCE(k) \
  if (BLOCK_SIZE >= k << 1) { \
    if (tid < k) temp[tid] = ::logsumexp2_fw_dev(temp[tid], temp[tid + k]); \
    __syncthreads(); \
  }
  REDUCE(512)
  REDUCE(256)
  REDUCE(128)
  REDUCE(64)
  REDUCE(32)
  REDUCE(16)
  REDUCE(8)
  REDUCE(4)
  REDUCE(2)
  REDUCE(1)
#undef REDUCE
  if (tid == 0) py[bid] = temp[0];
}

__global__ void broadcast_fw_dev(
    float *py, const float *px, unsigned skip1, unsigned skip2, unsigned size) {
  const unsigned i = IDX;
  if (i < size) py[i] = px[i % skip1 + (i / skip2) * skip1];
}

__global__ void batch_sum_fw_dev(
    float *py, const float *px, unsigned size, unsigned batch) {
  const unsigned i = IDX;
  if (i < size) {
    float temp = .0f;
    px += i;
    for (unsigned j = 0; j < batch; ++j, px += size) {
      temp += *px;
    }
    py[i] = temp;
  }
}

__global__ void add_grad_dev(
    float *pgx, const float *pgy, unsigned nx, unsigned ny) {
  const unsigned i = IDX;
  if (i < ::max(nx, ny)) ::atomicAdd(pgx + i % nx, pgy[i % ny]);
}

__global__ void add_grad_ofs_dev(
    float *pgx, const float *pgy,
    unsigned wx, unsigned wy, unsigned nx, unsigned ny) {
  const unsigned i = IDX;
  if (i < wy * ::max(nx, ny)) {
    ::atomicAdd(
        pgx + ((i / wy) * wx + (i % wy)) % (wx * nx),
        pgy[i % (wy * ny)]);
  }
}

__global__ void add_grad_sparse_dev(
    float *pgx, const float *pgy, unsigned wx, unsigned wy, unsigned repeat) {
  const unsigned i = IDX;
  if (i < wy * repeat) {
    ::atomicAdd(pgx + (i / wy) * wx + (i % wy), pgy[i]);
  }
}

#undef IDX

}  // namespace

namespace {

// Minimum requirements of the compute capability.
static const int MIN_CC_MAJOR = 3;
static const int MIN_CC_MINOR = 0;

}

namespace primitiv {

unsigned CUDADevice::num_devices() {
  int ret;
  CUDA_CALL(::hipGetDeviceCount(&ret));
  return ret;
}

void CUDADevice::initialize() {
  // Retrieves device properties.
  ::hipDeviceProp_t prop;
  CUDA_CALL(::hipGetDeviceProperties(&prop, dev_id_));

  // Dump device properties.
  cerr << "Selected CUDA Device " << dev_id_ << ':' << endl;
  cerr << "  Name ................. " << prop.name << endl;
  cerr << "  Global Memory ........ " << prop.totalGlobalMem << endl;
  cerr << "  Shared Memory ........ " << prop.sharedMemPerBlock << endl;
  cerr << "  Threads/block ........ " << prop.maxThreadsPerBlock << endl;
  cerr << "  Threads dim .......... " << prop.maxThreadsDim[0] << ','
                                      << prop.maxThreadsDim[1] << ','
                                      << prop.maxThreadsDim[2] << endl;
  cerr << "  Grid size ............ " << prop.maxGridSize[0] << ','
                                      << prop.maxGridSize[1] << ','
                                      << prop.maxGridSize[2] << endl;
  cerr << "  Compute Capability ... " << prop.major << '.'
                                      << prop.minor << endl;

  // Check compute capability requirements.
  if (prop.major < ::MIN_CC_MAJOR ||
      (prop.major == ::MIN_CC_MAJOR && prop.minor < ::MIN_CC_MINOR)) {
    THROW_ERROR(
        "CUDA Device " << dev_id_ << " does not satisfy the "
        "minimum requirement of the compute capability: "
        << prop.major << '.' << prop.minor << " < "
        << ::MIN_CC_MAJOR << '.' << ::MIN_CC_MINOR);
  }

  // Calculates size of dims to be used in CUDA kernels.
  dim1_x_ = 1;
  while (dim1_x_ < 1024 &&
      dim1_x_ < static_cast<unsigned>(prop.maxThreadsPerBlock)) {
    dim1_x_ <<= 1;
  }
  dim2_y_ = dim1_x_;
  dim2_x_ = 1;
  while (dim2_x_ < dim2_y_) {
    dim2_x_ <<= 1;
    dim2_y_ >>= 1;
  }
  cerr << "Block configuration:" << endl;
  cerr << "  1 dim .... " << dim1_x_ << " threads" << endl;
  cerr << "  2 dims ... " << dim2_x_ << "x" << dim2_y_ << " threads" << endl;

  // Initializes additional libraries
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUBLAS_CALL(::hipblasCreate(&cublas_));
  CURAND_CALL(::hiprandCreateGenerator(&curand_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(::hiprandSetPseudoRandomGeneratorSeed(curand_, rng_seed_));
}

CUDADevice::CUDADevice(unsigned device_id)
: dev_id_(device_id)
, rng_seed_(std::random_device()())
, pool_(device_id) {
  initialize();
}

CUDADevice::CUDADevice(unsigned device_id, unsigned rng_seed)
: dev_id_(device_id)
, rng_seed_(rng_seed)
, pool_(device_id) {
  initialize();
}

CUDADevice::~CUDADevice() {
  // Finalizes additional libraries
  CUBLAS_CALL(::hipblasDestroy(cublas_));
  CURAND_CALL(::hiprandDestroyGenerator(curand_));
}

std::shared_ptr<void> CUDADevice::new_handle(const Shape &shape) {
  return pool_.allocate(sizeof(float) * shape.size());
}

#define GRID_SIZE(x, threads) (((x) + (threads) - 1) / (threads))
#define DATA(x) static_cast<float *>((x).data())
#define CDATA(x) static_cast<const float *>((x).data())

std::vector<float> CUDADevice::tensor_to_vector_impl(const Tensor &x) {
  const unsigned size = x.shape().size();
  std::vector<float> ret(size);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        &ret[0], x.data(), sizeof(float) * size, hipMemcpyDeviceToHost));
  return ret;
}

void CUDADevice::reset_tensor_impl(Tensor &x, float k) {
  const unsigned size = x.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::set_const_dev<<<num_blocks, dim1_x_>>>(DATA(x), k, size);
}

void CUDADevice::reset_tensor_by_array_impl(Tensor &x, const float values[]) {
  const unsigned size = x.shape().size();
  CUDA_CALL(::hipSetDevice(dev_id_));
  CUDA_CALL(::hipMemcpy(
        x.data(), values, sizeof(float) * size, hipMemcpyHostToDevice));
}

void CUDADevice::copy_tensor_impl(const Tensor &x, Tensor &y) {
  switch (x.device()->type()) {
    case Device::DEVICE_TYPE_CPU:
      reset_tensor_by_array(y, CDATA(x));
      break;
    case Device::DEVICE_TYPE_CUDA:
      CUDA_CALL(::hipSetDevice(dev_id_));
      CUDA_CALL(::hipMemcpy(
            DATA(y), CDATA(x),
            sizeof(float) * x.shape().size(),
            hipMemcpyDeviceToDevice));
      break;
    default:
      reset_tensor_by_vector(y, x.to_vector());
  }
}

void CUDADevice::random_bernoulli_impl(float p, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(y), size));
  ::rand_bernoulli_dev<<<num_blocks, dim1_x_>>>(DATA(y), p, size);
}

void CUDADevice::random_uniform_impl(float lower, float upper, Tensor &y) {
  const unsigned size = y.shape().size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  const float scale = upper - lower;
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateUniform(curand_, DATA(y), size));
  ::rand_affine_dev<<<num_blocks, dim1_x_>>>(DATA(y), lower, scale, size);
}

void CUDADevice::random_normal_impl(float mean, float sd, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateNormal(
        curand_, DATA(y), y.shape().size(), mean, sd));
}

void CUDADevice::random_log_normal_impl(float mean, float sd, Tensor &y) {
  CUDA_CALL(::hipSetDevice(dev_id_));
  CURAND_CALL(::hiprandGenerateLogNormal(
        curand_, DATA(y), y.shape().size(), mean, sd));
}

Tensor CUDADevice::pick_fw_impl(
    const Tensor &x, unsigned dim,
    const std::vector<unsigned> &ids, Shape &&new_shape) {
  const unsigned base = new_shape.lower_volume(dim);
  const unsigned skip = base * x.shape()[dim];
  const unsigned size = new_shape.volume();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  const unsigned skip_x =
    (x.shape().has_batch()) * x.shape().volume();
  const unsigned skip_i = ids.size() > 1;
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  for (unsigned b = 0; b < new_shape.batch(); ++b) {
    ::slice_fw_dev<<<num_blocks, dim1_x_>>>(
        DATA(ret) + b * size, CDATA(x) + b * skip_x + base * ids[b * skip_i],
        base, skip, size);
  }
  return ret;
}

Tensor CUDADevice::slice_fw_impl(
    const Tensor &x, unsigned dim, unsigned offset, Shape &&new_shape) {
  const unsigned base = new_shape.lower_volume(dim);
  const unsigned span = base * new_shape[dim];
  const unsigned skip = base * x.shape()[dim];
  const unsigned size = new_shape.size();
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_);
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::slice_fw_dev<<<num_blocks, dim1_x_>>>(
      DATA(ret), CDATA(x) + base * offset, span, skip, size);
  return ret;
}

Tensor CUDADevice::concat_fw_impl(
    const std::vector<const Tensor *> &xs, unsigned dim, Shape &&new_shape) {
  const unsigned new_bs = new_shape.batch();
  const unsigned base = new_shape.lower_volume(dim);
  const unsigned skip = base * new_shape[dim];
  unsigned repeat = new_shape.volume() / skip;
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  unsigned offset = 0;
  for (const Tensor *x : xs) {
    const unsigned span = base * x->shape()[dim];
    const unsigned x_size = span * repeat * x->shape().batch();
    const unsigned y_size = span * repeat * new_bs;
    const unsigned num_blocks = GRID_SIZE(y_size, dim1_x_);
    ::concat_fw_dev<<<num_blocks, dim1_x_>>>(
        DATA(ret) + offset, CDATA(*x), span, skip, x_size, y_size);
    offset += span;
  }
  return ret;
}

#define CUDA_DEV_UNARY(name) \
Tensor CUDADevice::name##_fw_impl(const Tensor &x) { \
  Tensor ret = new_tensor(x.shape()); \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size, dim1_x_); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<num_blocks, dim1_x_>>>(DATA(ret), CDATA(x), size); \
  return ret; \
}

#define CUDA_DEV_BINARY_CONST(name) \
Tensor CUDADevice::name##_fw_impl(const Tensor &x, float k) { \
  Tensor ret = new_tensor(x.shape()); \
  const unsigned size = x.shape().size(); \
  const unsigned num_blocks = GRID_SIZE(size,dim1_x_); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<num_blocks, dim1_x_>>>(DATA(ret), CDATA(x), k, size); \
  return ret; \
}

#define CUDA_DEV_BINARY_SCALAR(name) \
Tensor CUDADevice::name##_fw_impl( \
    const Tensor &x, const Tensor &k, Shape &&new_shape) { \
  const unsigned size = new_shape.volume(); \
  const unsigned g1 = GRID_SIZE(size, dim1_x_); \
  const unsigned g2 = new_shape.batch(); \
  Tensor ret = new_tensor(new_shape); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<dim3(g1, g2, 1), dim1_x_>>>( \
      DATA(ret), CDATA(x), CDATA(k), size, \
      x.shape().has_batch(), k.shape().has_batch()); \
  return ret; \
}

#define CUDA_DEV_BINARY_AB(name) \
Tensor CUDADevice::name##_fw_impl( \
    const Tensor &a, const Tensor &b, Shape &&new_shape) { \
  const unsigned size = new_shape.volume(); \
  const unsigned x = GRID_SIZE(size, dim1_x_); \
  const unsigned y = new_shape.batch(); \
  Tensor ret = new_tensor(new_shape); \
  CUDA_CALL(::hipSetDevice(dev_id_)); \
  ::name##_fw_dev<<<dim3(x, y, 1), dim1_x_>>>( \
      DATA(ret), CDATA(a), CDATA(b), size, \
      a.shape().has_batch(), b.shape().has_batch()); \
  return ret; \
}

CUDA_DEV_UNARY(negate);
CUDA_DEV_UNARY(sqrt);
CUDA_DEV_UNARY(exp);
CUDA_DEV_UNARY(tanh);
CUDA_DEV_UNARY(sigmoid);
CUDA_DEV_UNARY(sin);
CUDA_DEV_UNARY(cos);
CUDA_DEV_UNARY(tan);

CUDA_DEV_BINARY_CONST(add_const);
CUDA_DEV_BINARY_CONST(subtract_const_r);
CUDA_DEV_BINARY_CONST(subtract_const_l);
CUDA_DEV_BINARY_CONST(multiply_const);
CUDA_DEV_BINARY_CONST(divide_const_r);
CUDA_DEV_BINARY_CONST(divide_const_l);
CUDA_DEV_BINARY_CONST(pstep);
CUDA_DEV_BINARY_CONST(prelu);

CUDA_DEV_BINARY_SCALAR(add_scalar);
CUDA_DEV_BINARY_SCALAR(subtract_scalar_r);
CUDA_DEV_BINARY_SCALAR(subtract_scalar_l);
CUDA_DEV_BINARY_SCALAR(multiply_scalar);
CUDA_DEV_BINARY_SCALAR(divide_scalar_r);
CUDA_DEV_BINARY_SCALAR(divide_scalar_l);

CUDA_DEV_BINARY_AB(add);
CUDA_DEV_BINARY_AB(subtract);
CUDA_DEV_BINARY_AB(multiply);
CUDA_DEV_BINARY_AB(divide);

#undef CUDA_DEV_UNARY
#undef CUDA_DEV_BINARY_CONST
#undef CUDA_DEV_BINARY_SCALAR
#undef CUDA_DEV_BINARY_AB

Tensor CUDADevice::transpose_fw_impl(const Tensor &x, Shape &&new_shape) {
  const unsigned d1 = new_shape[1];
  const unsigned d2 = new_shape[0];
  const unsigned bs = new_shape.batch();
  const unsigned g1 = GRID_SIZE(d1, dim2_x_);
  const unsigned g2 = GRID_SIZE(d2, dim2_y_);
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::transpose_fw_dev<<<dim3(g1, g2, bs), dim3(dim2_x_, dim2_y_, 1)>>>(
      DATA(ret), CDATA(x), d1, d2);
  return ret;
}

Tensor CUDADevice::matmul_fw_impl(
    const Tensor &a, const Tensor &b, Shape &&new_shape) {
  const unsigned di = new_shape[0];
  const unsigned dj = a.shape()[1];
  const unsigned dk = new_shape[1];
  float alpha = 1.;
  float beta = 0.;
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const unsigned a_skip = di * dj;
    const unsigned b_skip = b.shape().has_batch() * dj * dk;
    const unsigned y_skip = di * dk;
    const unsigned bs = a.shape().batch();
    for (unsigned n = 0; n < bs; ++n) {
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
            di, dk, dj,
            &alpha, CDATA(a) + n * a_skip, di, CDATA(b) + n * b_skip, dj,
            &beta, DATA(ret) + n * y_skip, di));
    }
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_N,
          di, dk * b.shape().batch(), dj,
          &alpha, CDATA(a), di, CDATA(b), dj,
          &beta, DATA(ret), di));
  }
  return ret;
}

void CUDADevice::matmul_bw_impl(
    const Tensor &a, const Tensor &b, const Tensor &gy,
    Tensor &ga, Tensor &gb) {
  // ga += gy . b^T
  // gb += a^T . gy
  const unsigned di = a.shape()[0];
  const unsigned dj = a.shape()[1];
  const unsigned dk = b.shape()[1];
  float alpha = 1.;
  float beta = 1.;
  CUDA_CALL(::hipSetDevice(dev_id_));
  if (a.shape().has_batch()) {
    // Do gemm multiple times.
    const unsigned a_skip = di * dj;
    const unsigned b_skip = b.shape().has_batch() * dj * dk;
    const unsigned y_skip = di * dk;
    const unsigned bs = a.shape().batch();
    for (unsigned n = 0; n < bs; ++n) {
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
            di, dj, dk,
            &alpha, CDATA(gy) + n * y_skip, di, CDATA(b) + n * b_skip, dj,
            &beta, DATA(ga) + n * a_skip, di));
      CUBLAS_CALL(::hipblasSgemm(
            cublas_, ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
            dj, dk, di,
            &alpha, CDATA(a) + n * a_skip, di, CDATA(gy) + n * y_skip, di,
            &beta, DATA(gb) + n * b_skip, dj));
    }
  } else {
    // Do gemm only once to calculate the product with a combined matrix.
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_N, ::HIPBLAS_OP_T,
          di, dj, dk * b.shape().batch(),
          &alpha, CDATA(gy), di, CDATA(b), dj,
          &beta, DATA(ga), di));
    CUBLAS_CALL(::hipblasSgemm(
          cublas_, ::HIPBLAS_OP_T, ::HIPBLAS_OP_N,
          dj, dk * b.shape().batch(), di,
          &alpha, CDATA(a), di, CDATA(gy), di,
          &beta, DATA(gb), dj));
  }
}

Tensor CUDADevice::sum_fw_impl(const Tensor &x, unsigned dim) {
  const Shape new_shape = x.shape().resize_dim(dim, 1);
  const unsigned n = x.shape()[dim];
  const unsigned r = new_shape.size();
  const unsigned s = new_shape.lower_volume(dim);
  unsigned block_size = dim1_x_;
  while (block_size >> 1 >= n) block_size >>= 1;
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  switch (block_size) {
#define CASE(k) \
    case k: ::sum_fw_dev<k><<<r, k>>>(DATA(ret), CDATA(x), s, n); break
    CASE(1024);
    CASE(512);
    CASE(256);
    CASE(128);
    CASE(64);
    CASE(32);
    CASE(16);
    CASE(8);
    CASE(4);
    CASE(2);
    CASE(1);
#undef CASE
  }
  return ret;
}

Tensor CUDADevice::logsumexp_fw_impl(const Tensor &x, unsigned dim) {
  const Shape new_shape = x.shape().resize_dim(dim, 1);
  const unsigned n = x.shape()[dim];
  const unsigned r = new_shape.size();
  const unsigned s = new_shape.lower_volume(dim);
  unsigned block_size = dim1_x_;
  while (block_size >> 1 >= n) block_size >>= 1;
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  switch (block_size) {
#define CASE(k) \
    case k: ::logsumexp_fw_dev<k><<<r, k>>>(DATA(ret), CDATA(x), s, n); break
    CASE(1024);
    CASE(512);
    CASE(256);
    CASE(128);
    CASE(64);
    CASE(32);
    CASE(16);
    CASE(8);
    CASE(4);
    CASE(2);
    CASE(1);
#undef CASE
  }
  return ret;
}

Tensor CUDADevice::broadcast_fw_impl(
    const Tensor &x, unsigned dim, unsigned size, Shape &&new_shape) {
  const unsigned skip1 = new_shape.lower_volume(dim);
  const unsigned skip2 = skip1 * size;
  const unsigned total = new_shape.size();
  const unsigned g1 = GRID_SIZE(total, dim1_x_);
  Tensor ret = new_tensor(new_shape);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::broadcast_fw_dev<<<g1, dim1_x_>>>(DATA(ret), CDATA(x), skip1, skip2, total);
  return ret;
}

Tensor CUDADevice::batch_sum_fw_impl(const Tensor &x) {
  Tensor ret = new_tensor(x.shape().resize_batch(1));
  const unsigned size = ret.shape().size();
  const unsigned g1 = GRID_SIZE(size, dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::batch_sum_fw_dev<<<g1, dim1_x_>>>(
      DATA(ret), CDATA(x), size, x.shape().batch());
  return ret;
}

void CUDADevice::add_gradient_impl(Tensor &a, const Tensor &b) {
  const unsigned nx = a.shape().size();
  const unsigned ny = b.shape().size();
  const unsigned g1 = GRID_SIZE(std::max(nx, ny), dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::add_grad_dev<<<g1, dim1_x_>>>(DATA(a), CDATA(b), nx, ny);
}

void CUDADevice::add_gradient_offset_impl(
    Tensor &a, const Tensor &b, unsigned dim, unsigned offset) {
  const Shape &sa = a.shape();
  const Shape &sb = b.shape();
  const unsigned base = sa.lower_volume(dim);
  unsigned repeat = 1;
  for (unsigned i = dim + 1; i < sa.depth(); ++i) repeat *= sa[i];
  const unsigned ox = base * offset;
  const unsigned wx = base * sa[dim];
  const unsigned wy = base * sb[dim];
  const unsigned nx = repeat * sa.batch();
  const unsigned ny = repeat * sb.batch();
  const unsigned g1 = GRID_SIZE(wy * std::max(nx, ny), dim1_x_);
  CUDA_CALL(::hipSetDevice(dev_id_));
  ::add_grad_ofs_dev<<<g1, dim1_x_>>>(DATA(a) + ox, CDATA(b), wx, wy, nx, ny);
}

void CUDADevice::add_gradient_sparse_impl(
    Tensor &a, const Tensor &b,
    unsigned dim, const std::vector<unsigned>& ids) {
  const Shape &sa = a.shape();
  const Shape &sb = b.shape();
  const unsigned size = sb.volume();
  const unsigned base = sb.lower_volume(dim);
  const unsigned repeat = size / base;
  const unsigned wx = base * sa[dim];
  const unsigned g1 = GRID_SIZE(size, dim1_x_);
  const unsigned bs = sb.batch();
  const unsigned skip_a = (sa.has_batch()) * sa.volume();
  const unsigned skip_i = ids.size() > 1;
  float *dest = DATA(a);
  const float *src = CDATA(b);

  CUDA_CALL(::hipSetDevice(dev_id_));
  for (unsigned batch = 0; batch < bs; ++batch) {
    ::add_grad_sparse_dev<<<g1, dim1_x_>>>(
        dest + batch * skip_a + base * ids[batch * skip_i],
        src + batch * size,
        wx, base, repeat);
  }
}

}  // namespace primitiv
